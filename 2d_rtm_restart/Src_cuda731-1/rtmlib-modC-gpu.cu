#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gpu.h"


//propagation data
 float *g_Vx0;
 float *g_Vz0;
 float *g_sigmaxx0; 
 float *g_sigmazz0;
 float *g_sigmaxz0;


//extended propagation data residence in GPU device
__device__ float *ex_aux_m2m3_c;
__device__ float *ex_aux_m2_c; 
__device__ float *ex_aux_m3_c; 
__device__ float *ex_sigmaxx0; 
__device__ float *ex_sigmazz0; 
__device__ float *ex_sigmaxz0; 
__device__ float *ex_Vx0;
__device__ float *ex_Vz0;
__device__ float *ex_m1_x; 
__device__ float *ex_m1_z; 



//constant data, extended, with 10 more layers over the CPU version 
 float *g_ex_m1_x;
 float *g_ex_m1_z;
 float *g_ex_aux_m2_c; 
 float *g_ex_aux_m3_c; 
 float *g_ex_aux_m2m3_c; 

__global__ void rtm_gpu_kernel(int it,int nt, int nz, int nx,
        float * g_Vx0, float * g_Vz0, float * g_sigmaxx0, float * g_sigmazz0, float * g_sigmaxz0, //(nz, nx, nt)
        float * g_ex_m1_x,float * g_ex_m1_z,float * g_ex_aux_m2_c, float * g_ex_aux_m3_c, float * g_ex_aux_m2m3_c);//(nz+10,	nx+10)


extern "C" void rtm_gpu_init(int nt, int nz, int nx) 
{
	//set cuda devices and put all data onto gpu memory
	
	hipError_t cuda_ret;
     	hipError_t err;

	//Set Device 
    	cuda_ret = hipSetDevice(0);
	if(cuda_ret != hipSuccess){
		fprintf(stderr, "Failed to Set The cuda Device !\n");
	}
	else{
		fprintf(stderr, "GPU Device Set ====> OK\n");
	}

	// data init
	hipMalloc(&g_Vx0,sizeof(float)*nx*nz*nt);
	hipMalloc(&g_Vz0,sizeof(float)*nx*nz*nt);
	hipMalloc(&g_sigmaxx0,sizeof(float)*nx*nz*nt);
	hipMalloc(&g_sigmazz0,sizeof(float)*nx*nz*nt);
	hipMalloc(&g_sigmaxz0,sizeof(float)*nx*nz*nt);
	hipMalloc(&g_ex_m1_x,sizeof(float)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_m1_z,sizeof(float)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_aux_m2_c,sizeof(float)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_aux_m3_c,sizeof(float)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_aux_m2m3_c,sizeof(float)*(nx+10)*(nz+10));


	hipFuncSetCacheConfig(reinterpret_cast<const void*>(rtm_gpu_kernel),hipFuncCachePreferL1);

	fprintf(stderr,"GPU Data Init ====> OK\n");

	// data copy

//	hipMemcpy(g_Vx0, Vx0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
//	hipMemcpy(g_Vz0, Vz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
//	hipMemcpy(g_sigmaxx0, sigmaxx0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
//	hipMemcpy(g_sigmaxz0, sigmaxz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
//	hipMemcpy(g_sigmazz0, sigmazz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
//	hipMemcpy(g_m1_x, m1_x, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
//	hipMemcpy(g_m1_z, m1_z, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
//	hipMemcpy(g_aux_m2_c, aux_m2_c, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
//	hipMemcpy(g_aux_m3_c, aux_m3_c, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
//	hipMemcpy(g_aux_m2m3_c, aux_m2m3_c, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
//	fprintf(stderr,"Data Copy To GPU OK\n");
}



extern "C" void rtm_gpu_copy_in(int nt, int nz, int nx, 
        float * Vx0, float * Vz0, float * sigmaxx0, float * sigmazz0, float * sigmaxz0, //(nz, nx, nt)
        float * ex_m1_x,float * ex_m1_z,float * ex_aux_m2_c, float * ex_aux_m3_c, float * ex_aux_m2m3_c)//(nz,	nx)
{
	// data copy

	hipMemcpy(g_Vx0, Vx0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
	hipMemcpy(g_Vz0, Vz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
	hipMemcpy(g_sigmaxx0, sigmaxx0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
	hipMemcpy(g_sigmaxz0, sigmaxz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
	hipMemcpy(g_sigmazz0, sigmazz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
	hipMemcpy(g_ex_m1_x, ex_m1_x, sizeof(float)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_m1_z, ex_m1_z, sizeof(float)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_aux_m2_c, ex_aux_m2_c, sizeof(float)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_aux_m3_c, ex_aux_m3_c, sizeof(float)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_aux_m2m3_c, ex_aux_m2m3_c, sizeof(float)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	
	fprintf(stderr,"Copy out for debuging \n");
	hipMemcpy(ex_m1_x, g_ex_m1_x, sizeof(float)*(nx+10)*(nz+10), hipMemcpyDeviceToHost);
	hipMemcpy(ex_m1_z, g_ex_m1_z, sizeof(float)*(nx+10)*(nz+10), hipMemcpyDeviceToHost);
	hipMemcpy(ex_aux_m2_c, g_ex_aux_m2_c, sizeof(float)*(nx+10)*(nz+10), hipMemcpyDeviceToHost);
	hipMemcpy(ex_aux_m3_c, g_ex_aux_m3_c, sizeof(float)*(nx+10)*(nz+10), hipMemcpyDeviceToHost);
	hipMemcpy(ex_aux_m2m3_c, g_ex_aux_m2m3_c, sizeof(float)*(nx+10)*(nz+10), hipMemcpyDeviceToHost);
	
	fprintf(stderr,"Data Copy To GPU  ====> OK\n");
}

extern "C" void rtm_gpu_copy_out_debug(int nt, int nz, int nx, 
        float * Vx0, float * Vz0, float * sigmaxx0, float * sigmazz0, float * sigmaxz0, //(nz, nx, nt)
        float * ex_m1_x,float * ex_m1_z,float * ex_aux_m2_c, float * ex_aux_m3_c, float * ex_aux_m2m3_c)//(nz,	nx)
{
	// data copy

//	hipMemcpy(g_Vx0, Vx0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
//	hipMemcpy(g_Vz0, Vz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
//	hipMemcpy(g_sigmaxx0, sigmaxx0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
//	hipMemcpy(g_sigmaxz0, sigmaxz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
//	hipMemcpy(g_sigmazz0, sigmazz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
//	hipMemcpy(g_ex_m1_x, ex_m1_x, sizeof(float)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
//	hipMemcpy(g_ex_m1_z, ex_m1_z, sizeof(float)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
//	hipMemcpy(g_ex_aux_m2_c, ex_aux_m2_c, sizeof(float)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
//	hipMemcpy(g_ex_aux_m3_c, ex_aux_m3_c, sizeof(float)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
//	hipMemcpy(g_ex_aux_m2m3_c, ex_aux_m2m3_c, sizeof(float)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	
	fprintf(stderr,"Copy out for debuging \n");
	hipMemcpy(ex_m1_x, g_ex_m1_x, sizeof(float)*(nx+10)*(nz+10), hipMemcpyDeviceToHost);
	hipMemcpy(ex_m1_z, g_ex_m1_z, sizeof(float)*(nx+10)*(nz+10), hipMemcpyDeviceToHost);
	hipMemcpy(ex_aux_m2_c, g_ex_aux_m2_c, sizeof(float)*(nx+10)*(nz+10), hipMemcpyDeviceToHost);
	hipMemcpy(ex_aux_m3_c, g_ex_aux_m3_c, sizeof(float)*(nx+10)*(nz+10), hipMemcpyDeviceToHost);
	hipMemcpy(ex_aux_m2m3_c, g_ex_aux_m2m3_c, sizeof(float)*(nx+10)*(nz+10), hipMemcpyDeviceToHost);
	
}


extern "C" void rtm_gpu_copy_out(int nt, int nz, int nx, 
        float * Vx0, float * Vz0, float * sigmaxx0, float * sigmazz0, float * sigmaxz0)//, //(nz, nx, nt)
{
	// data copy back from GPU mem
	hipMemcpy(Vx0, g_Vx0, sizeof(float)*nx*nz*nt,  		hipMemcpyDeviceToHost);
	hipMemcpy(Vz0, g_Vz0,sizeof(float)*nx*nz*nt, 			hipMemcpyDeviceToHost);
	hipMemcpy(sigmaxx0, g_sigmaxx0, sizeof(float)*nx*nz*nt, 	hipMemcpyDeviceToHost);
	hipMemcpy(sigmaxz0, g_sigmaxz0, sizeof(float)*nx*nz*nt, 	hipMemcpyDeviceToHost);
	hipMemcpy(sigmazz0, g_sigmazz0,  sizeof(float)*nx*nz*nt, 	hipMemcpyDeviceToHost);
	//hipMemcpy(sigmazz0, g_sigmazz0,  sizeof(float)*nx*nz*nt, 	hipMemcpyDeviceToHost);
	fprintf(stderr,"Data Copy To CPU ====> OK\n");

}


extern "C" void rtm_gpu_final()
{

	//release GPU memory space
	hipFree(&g_Vx0);
	hipFree(&g_Vz0);
	hipFree(&g_sigmaxx0);
	hipFree(&g_sigmazz0);
	hipFree(&g_sigmaxz0);
	hipFree(&g_ex_m1_x);
	hipFree(&g_ex_m1_z);
	hipFree(&g_ex_aux_m2_c);
	hipFree(&g_ex_aux_m3_c);
	hipFree(&g_ex_aux_m2m3_c);
	fprintf(stderr,"GPU Mem Released ====> OK\n");
}


__global__ void rtm_gpu_kernel(int it,int nt, int nz, int nx,
        float * g_Vx0, float * g_Vz0, float * g_sigmaxx0, float * g_sigmazz0, float * g_sigmaxz0, //(nz, nx, nt)
        float * g_ex_m1_x,float * g_ex_m1_z,float * g_ex_aux_m2_c, float * g_ex_aux_m3_c, float * g_ex_aux_m2m3_c)//(nz+10,	nx+10)
{

	float c1=35.0/294912.0,c2=-405.0/229376.0,c3=567.0/40960.0,c4=-735.0/8192.0,c5=19845.0/16384.0;

	//GPU thread index
	int iz, ix;
	iz = blockIdx.x*blockDim.x + threadIdx.x;
	ix = blockIdx.y*blockDim.y + threadIdx.y;
	//gt = it;
 	
              g_Vx0[index3d(iz,ix  ,it)] = g_Vx0[index3d(iz,ix  ,it)]	+ g_Vx0[index3d(iz, ix, it+2)]
									+ g_ex_aux_m2m3_c[index_ex(iz,ix-5)]*c1*g_sigmaxx0[index3d(iz,ix-5,it+1)]							
							 		+ g_ex_aux_m2m3_c[index_ex(iz,ix-4)]*c2*g_sigmaxx0[index3d(iz,ix-4,it+1)]		
									+ g_ex_aux_m2m3_c[index_ex(iz,ix-3)]*c3*g_sigmaxx0[index3d(iz,ix-3,it+1)]	
									+ g_ex_aux_m2m3_c[index_ex(iz,ix-2)]*c4*g_sigmaxx0[index3d(iz,ix-2,it+1)]	
									+ g_ex_aux_m2m3_c[index_ex(iz,ix-1)]*c5*g_sigmaxx0[index3d(iz,ix-1,it+1)]	
									- g_ex_aux_m2m3_c[index_ex(iz,ix)]  *c5*g_sigmaxx0[index3d(iz,ix,it+1)]	
									- g_ex_aux_m2m3_c[index_ex(iz,ix+1)]*c4*g_sigmaxx0[index3d(iz,ix+1,it+1)]	
									- g_ex_aux_m2m3_c[index_ex(iz,ix+2)]*c3*g_sigmaxx0[index3d(iz,ix+2,it+1)]	
									- g_ex_aux_m2m3_c[index_ex(iz,ix+3)]*c2*g_sigmaxx0[index3d(iz,ix+3,it+1)]	
									- g_ex_aux_m2m3_c[index_ex(iz,ix+4)]*c1*g_sigmaxx0[index3d(iz,ix+4,it+1)]


									+ g_ex_aux_m2_c[index_ex(iz,ix-5)]*c1*g_sigmazz0[index3d(iz,ix-5,it+1)]							
							 		+ g_ex_aux_m2_c[index_ex(iz,ix-4)]*c2*g_sigmazz0[index3d(iz,ix-4,it+1)]		
									+ g_ex_aux_m2_c[index_ex(iz,ix-3)]*c3*g_sigmazz0[index3d(iz,ix-3,it+1)]	
									+ g_ex_aux_m2_c[index_ex(iz,ix-2)]*c4*g_sigmazz0[index3d(iz,ix-2,it+1)]	
									+ g_ex_aux_m2_c[index_ex(iz,ix-1)]*c5*g_sigmazz0[index3d(iz,ix-1,it+1)]	
									- g_ex_aux_m2_c[index_ex(iz,ix)]  *c5*g_sigmazz0[index3d(iz,ix,it+1)]	
									- g_ex_aux_m2_c[index_ex(iz,ix+1)]*c4*g_sigmazz0[index3d(iz,ix+1,it+1)]	
									- g_ex_aux_m2_c[index_ex(iz,ix+2)]*c3*g_sigmazz0[index3d(iz,ix+2,it+1)]	
									- g_ex_aux_m2_c[index_ex(iz,ix+3)]*c2*g_sigmazz0[index3d(iz,ix+3,it+1)]	
									- g_ex_aux_m2_c[index_ex(iz,ix+4)]*c1*g_sigmazz0[index3d(iz,ix+4,it+1)]	
	


									+ g_ex_aux_m3_c[index_ex(iz-4,ix)]*c1*g_sigmaxz0[index3d(iz-4,ix,it+1)]		
									+ g_ex_aux_m3_c[index_ex(iz-3,ix)]*c2*g_sigmaxz0[index3d(iz-3,ix,it+1)]	
									+ g_ex_aux_m3_c[index_ex(iz-2,ix)]*c3*g_sigmaxz0[index3d(iz-2,ix,it+1)]	
									+ g_ex_aux_m3_c[index_ex(iz-1,ix)]*c4*g_sigmaxz0[index3d(iz-1,ix,it+1)]	
									+ g_ex_aux_m3_c[index_ex(iz,ix)]  *c5*g_sigmaxz0[index3d(iz,ix,it+1)]	
									- g_ex_aux_m3_c[index_ex(iz+1,ix)]*c5*g_sigmaxz0[index3d(iz+1,ix,it+1)]	
									- g_ex_aux_m3_c[index_ex(iz+2,ix)]*c4*g_sigmaxz0[index3d(iz+2,ix,it+1)]	
									- g_ex_aux_m3_c[index_ex(iz+3,ix)]*c3*g_sigmaxz0[index3d(iz+3,ix,it+1)]	
									- g_ex_aux_m3_c[index_ex(iz+4,ix)]*c2*g_sigmaxz0[index3d(iz+4,ix,it+1)]	
									- g_ex_aux_m3_c[index_ex(iz+5,ix)]*c1*g_sigmaxz0[index3d(iz+5,ix,it+1)]	;						

 
     __syncthreads();

            g_Vz0[index3d(iz,ix  ,it)] = g_Vz0[index3d(iz,ix  ,it)]  	+ g_Vz0[index3d(iz,ix  ,it+2)] 
	     								+ g_ex_aux_m2_c[index_ex(iz-5,ix)]*c1*g_sigmaxx0[index3d(iz-5,ix,it+1)]							
	     						 		+ g_ex_aux_m2_c[index_ex(iz-4,ix)]*c2*g_sigmaxx0[index3d(iz-4,ix,it+1)]		
	     								+ g_ex_aux_m2_c[index_ex(iz-3,ix)]*c3*g_sigmaxx0[index3d(iz-3,ix,it+1)]	
	     								+ g_ex_aux_m2_c[index_ex(iz-2,ix)]*c4*g_sigmaxx0[index3d(iz-2,ix,it+1)]	
	     								+ g_ex_aux_m2_c[index_ex(iz-1,ix)]*c5*g_sigmaxx0[index3d(iz-1,ix,it+1)]	
	     								- g_ex_aux_m2_c[index_ex(iz,ix)]  *c5*g_sigmaxx0[index3d(iz,ix,it+1)]	
	     								- g_ex_aux_m2_c[index_ex(iz+1,ix)]*c4*g_sigmaxx0[index3d(iz+1,ix,it+1)]	
	     								- g_ex_aux_m2_c[index_ex(iz+2,ix)]*c3*g_sigmaxx0[index3d(iz+2,ix,it+1)]	
	     								- g_ex_aux_m2_c[index_ex(iz+3,ix)]*c2*g_sigmaxx0[index3d(iz+3,ix,it+1)]	
	     								- g_ex_aux_m2_c[index_ex(iz+4,ix)]*c1*g_sigmaxx0[index3d(iz+4,ix,it+1)]	
	     
	
	             							+ g_ex_aux_m2m3_c[index_ex(iz-5,ix)]*c1*g_sigmazz0[index3d(iz-5,ix,it+1)]							
	     						 		+ g_ex_aux_m2m3_c[index_ex(iz-4,ix)]*c2*g_sigmazz0[index3d(iz-4,ix,it+1)]		
	     								+ g_ex_aux_m2m3_c[index_ex(iz-3,ix)]*c3*g_sigmazz0[index3d(iz-3,ix,it+1)]	
	     								+ g_ex_aux_m2m3_c[index_ex(iz-2,ix)]*c4*g_sigmazz0[index3d(iz-2,ix,it+1)]	
	     								+ g_ex_aux_m2m3_c[index_ex(iz-1,ix)]*c5*g_sigmazz0[index3d(iz-1,ix,it+1)]	
	     								- g_ex_aux_m2m3_c[index_ex(iz,ix)]  *c5*g_sigmazz0[index3d(iz,ix,it+1)]	
	     								- g_ex_aux_m2m3_c[index_ex(iz+1,ix)]*c4*g_sigmazz0[index3d(iz+1,ix,it+1)]	
	     								- g_ex_aux_m2m3_c[index_ex(iz+2,ix)]*c3*g_sigmazz0[index3d(iz+2,ix,it+1)]	
	     								- g_ex_aux_m2m3_c[index_ex(iz+3,ix)]*c2*g_sigmazz0[index3d(iz+3,ix,it+1)]	
	     								- g_ex_aux_m2m3_c[index_ex(iz+4,ix)]*c1*g_sigmazz0[index3d(iz+4,ix,it+1)]	
	     
	     								+ g_ex_aux_m3_c[index_ex(iz,ix-4)]*c1*g_sigmaxz0[index3d(iz,ix-4,it+1)]		
	     								+ g_ex_aux_m3_c[index_ex(iz,ix-3)]*c2*g_sigmaxz0[index3d(iz,ix-3,it+1)]	
	     								+ g_ex_aux_m3_c[index_ex(iz,ix-2)]*c3*g_sigmaxz0[index3d(iz,ix-2,it+1)]	
	     								+ g_ex_aux_m3_c[index_ex(iz,ix-1)]*c4*g_sigmaxz0[index3d(iz,ix-1,it+1)]	
	     								+ g_ex_aux_m3_c[index_ex(iz,ix)]  *c5*g_sigmaxz0[index3d(iz,ix,it+1)]	
	     								- g_ex_aux_m3_c[index_ex(iz,ix+1)]*c5*g_sigmaxz0[index3d(iz,ix+1,it+1)]	
	     								- g_ex_aux_m3_c[index_ex(iz,ix+2)]*c4*g_sigmaxz0[index3d(iz,ix+2,it+1)]	
	     								- g_ex_aux_m3_c[index_ex(iz,ix+3)]*c3*g_sigmaxz0[index3d(iz,ix+3,it+1)]	
	     								- g_ex_aux_m3_c[index_ex(iz,ix+4)]*c2*g_sigmaxz0[index3d(iz,ix+4,it+1)]	
	     								- g_ex_aux_m3_c[index_ex(iz,ix+5)]*c1*g_sigmaxz0[index3d(iz,ix+5,it+1)]	;							
	
              g_sigmaxx0[index3d(iz,ix  ,it)] = g_sigmaxx0[index3d(iz,ix  ,it)]	+ g_sigmaxx0[index3d(iz,ix  ,it+2)] 
        									+ g_ex_m1_x[index_ex(iz,ix-4)]*c1*g_Vx0[index3d(iz,ix-4,it+1)]		
        									+ g_ex_m1_x[index_ex(iz,ix-3)]*c2*g_Vx0[index3d(iz,ix-3,it+1)]	
        									+ g_ex_m1_x[index_ex(iz,ix-2)]*c3*g_Vx0[index3d(iz,ix-2,it+1)]	
        									+ g_ex_m1_x[index_ex(iz,ix-1)]*c4*g_Vx0[index3d(iz,ix-1,it+1)]	
        									+ g_ex_m1_x[index_ex(iz,ix)]  *c5*g_Vx0[index3d(iz,ix,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+1)]*c5*g_Vx0[index3d(iz,ix+1,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+2)]*c4*g_Vx0[index3d(iz,ix+2,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+3)]*c3*g_Vx0[index3d(iz,ix+3,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+4)]*c2*g_Vx0[index3d(iz,ix+4,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+5)]*c1*g_Vx0[index3d(iz,ix+5,it+1)]	;						
 
    __syncthreads();
             g_sigmazz0[index3d(iz,ix  ,it)] = g_sigmazz0[index3d(iz,ix  ,it)]	+ g_sigmazz0[index3d(iz,ix  ,it+2)] 
										+ g_ex_m1_z[index_ex(iz-4,ix)]*c1*g_Vz0[index3d(iz-4,ix,it+1)]		
										+ g_ex_m1_z[index_ex(iz-3,ix)]*c2*g_Vz0[index3d(iz-3,ix,it+1)]	
										+ g_ex_m1_z[index_ex(iz-2,ix)]*c3*g_Vz0[index3d(iz-2,ix,it+1)]	
										+ g_ex_m1_z[index_ex(iz-1,ix)]*c4*g_Vz0[index3d(iz-1,ix,it+1)]	
										+ g_ex_m1_z[index_ex(iz,ix)]  *c5*g_Vz0[index3d(iz,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+1,ix)]*c5*g_Vz0[index3d(iz+1,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+2,ix)]*c4*g_Vz0[index3d(iz+2,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+3,ix)]*c3*g_Vz0[index3d(iz+3,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+4,ix)]*c2*g_Vz0[index3d(iz+4,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+5,ix)]*c1*g_Vz0[index3d(iz+5,ix,it+1)]	;						
     __syncthreads();
     g_sigmaxz0[index3d(iz,ix  ,it)] = g_sigmaxz0[index3d(iz,ix  ,it)]	+ g_sigmaxz0[index3d(iz,ix  ,it+2)]	 
										+ g_ex_m1_x[index_ex(iz-5,ix)]*c1*g_Vx0[index3d(iz-5,ix,it+1)]							
							 			+ g_ex_m1_x[index_ex(iz-4,ix)]*c2*g_Vx0[index3d(iz-4,ix,it+1)]		
										+ g_ex_m1_x[index_ex(iz-3,ix)]*c3*g_Vx0[index3d(iz-3,ix,it+1)]	
										+ g_ex_m1_x[index_ex(iz-2,ix)]*c4*g_Vx0[index3d(iz-2,ix,it+1)]	
										+ g_ex_m1_x[index_ex(iz-1,ix)]*c5*g_Vx0[index3d(iz-1,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz,ix)]  *c5*g_Vx0[index3d(iz,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+1,ix)]*c4*g_Vx0[index3d(iz+1,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+2,ix)]*c3*g_Vx0[index3d(iz+2,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+3,ix)]*c2*g_Vx0[index3d(iz+3,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+4,ix)]*c1*g_Vx0[index3d(iz+4,ix,it+1)]	//;
	
        
										+ g_ex_m1_z[index_ex(iz,ix-5)]*c1*g_Vz0[index3d(iz,ix-5,it+1)]							
							 			+ g_ex_m1_z[index_ex(iz,ix-4)]*c2*g_Vz0[index3d(iz,ix-4,it+1)]		
										+ g_ex_m1_z[index_ex(iz,ix-3)]*c3*g_Vz0[index3d(iz,ix-3,it+1)]	
										+ g_ex_m1_z[index_ex(iz,ix-2)]*c4*g_Vz0[index3d(iz,ix-2,it+1)]	
										+ g_ex_m1_z[index_ex(iz,ix-1)]*c5*g_Vz0[index3d(iz,ix-1,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix)]  *c5*g_Vz0[index3d(iz,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+1)]*c4*g_Vz0[index3d(iz,ix+1,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+2)]*c3*g_Vz0[index3d(iz,ix+2,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+3)]*c2*g_Vz0[index3d(iz,ix+3,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+4)]*c1*g_Vz0[index3d(iz,ix+4,it+1)]	;
		
	__syncthreads();


	}

__global__ void rtm_gpu_kernelB(int it,int nt, int nz, int nx,
        float * g_Vx0, float * g_Vz0, float * g_sigmaxx0, float * g_sigmazz0, float * g_sigmaxz0, //(nz, nx, nt)
        float * g_ex_m1_x,float * g_ex_m1_z,float * g_ex_aux_m2_c, float * g_ex_aux_m3_c, float * g_ex_aux_m2m3_c)//(nz+10,	nx+10)
{

	float c1=35.0/294912.0,c2=-405.0/229376.0,c3=567.0/40960.0,c4=-735.0/8192.0,c5=19845.0/16384.0;

	int iz, ix;
	iz = blockIdx.x*blockDim.x + threadIdx.x;
	ix = blockIdx.y*blockDim.y + threadIdx.y;


              g_sigmaxx0[index3d(iz,ix  ,it)] = g_sigmaxx0[index3d(iz,ix  ,it)]	+ g_sigmaxx0[index3d(iz,ix  ,it+2)] 
        									+ g_ex_m1_x[index_ex(iz,ix-4)]*c1*g_Vx0[index3d(iz,ix-4,it+1)]		
        									+ g_ex_m1_x[index_ex(iz,ix-3)]*c2*g_Vx0[index3d(iz,ix-3,it+1)]	
        									+ g_ex_m1_x[index_ex(iz,ix-2)]*c3*g_Vx0[index3d(iz,ix-2,it+1)]	
        									+ g_ex_m1_x[index_ex(iz,ix-1)]*c4*g_Vx0[index3d(iz,ix-1,it+1)]	
        									+ g_ex_m1_x[index_ex(iz,ix)]  *c5*g_Vx0[index3d(iz,ix,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+1)]*c5*g_Vx0[index3d(iz,ix+1,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+2)]*c4*g_Vx0[index3d(iz,ix+2,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+3)]*c3*g_Vx0[index3d(iz,ix+3,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+4)]*c2*g_Vx0[index3d(iz,ix+4,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+5)]*c1*g_Vx0[index3d(iz,ix+5,it+1)]	;						
     __syncthreads();
             g_sigmazz0[index3d(iz,ix  ,it)] = g_sigmazz0[index3d(iz,ix  ,it)]	+ g_sigmazz0[index3d(iz,ix  ,it+2)] 
										+ g_ex_m1_z[index_ex(iz-4,ix)]*c1*g_Vz0[index3d(iz-4,ix,it+1)]		
										+ g_ex_m1_z[index_ex(iz-3,ix)]*c2*g_Vz0[index3d(iz-3,ix,it+1)]	
										+ g_ex_m1_z[index_ex(iz-2,ix)]*c3*g_Vz0[index3d(iz-2,ix,it+1)]	
										+ g_ex_m1_z[index_ex(iz-1,ix)]*c4*g_Vz0[index3d(iz-1,ix,it+1)]	
										+ g_ex_m1_z[index_ex(iz,ix)]  *c5*g_Vz0[index3d(iz,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+1,ix)]*c5*g_Vz0[index3d(iz+1,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+2,ix)]*c4*g_Vz0[index3d(iz+2,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+3,ix)]*c3*g_Vz0[index3d(iz+3,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+4,ix)]*c2*g_Vz0[index3d(iz+4,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+5,ix)]*c1*g_Vz0[index3d(iz+5,ix,it+1)]	;						
     __syncthreads();
     g_sigmaxz0[index3d(iz,ix  ,it)] = g_sigmaxz0[index3d(iz,ix  ,it)]	+ g_sigmaxz0[index3d(iz,ix  ,it+2)]	 
										+ g_ex_m1_x[index_ex(iz-5,ix)]*c1*g_Vx0[index3d(iz-5,ix,it+1)]							
							 			+ g_ex_m1_x[index_ex(iz-4,ix)]*c2*g_Vx0[index3d(iz-4,ix,it+1)]		
										+ g_ex_m1_x[index_ex(iz-3,ix)]*c3*g_Vx0[index3d(iz-3,ix,it+1)]	
										+ g_ex_m1_x[index_ex(iz-2,ix)]*c4*g_Vx0[index3d(iz-2,ix,it+1)]	
										+ g_ex_m1_x[index_ex(iz-1,ix)]*c5*g_Vx0[index3d(iz-1,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz,ix)]  *c5*g_Vx0[index3d(iz,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+1,ix)]*c4*g_Vx0[index3d(iz+1,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+2,ix)]*c3*g_Vx0[index3d(iz+2,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+3,ix)]*c2*g_Vx0[index3d(iz+3,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+4,ix)]*c1*g_Vx0[index3d(iz+4,ix,it+1)]	//;
	
        
      //sigmaxz0[index3d(iz,ix  ,it)] = sigmaxz0[index3d(iz,ix  ,it)]	
										+ g_ex_m1_z[index_ex(iz,ix-5)]*c1*g_Vz0[index3d(iz,ix-5,it+1)]							
							 			+ g_ex_m1_z[index_ex(iz,ix-4)]*c2*g_Vz0[index3d(iz,ix-4,it+1)]		
										+ g_ex_m1_z[index_ex(iz,ix-3)]*c3*g_Vz0[index3d(iz,ix-3,it+1)]	
										+ g_ex_m1_z[index_ex(iz,ix-2)]*c4*g_Vz0[index3d(iz,ix-2,it+1)]	
										+ g_ex_m1_z[index_ex(iz,ix-1)]*c5*g_Vz0[index3d(iz,ix-1,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix)]  *c5*g_Vz0[index3d(iz,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+1)]*c4*g_Vz0[index3d(iz,ix+1,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+2)]*c3*g_Vz0[index3d(iz,ix+2,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+3)]*c2*g_Vz0[index3d(iz,ix+3,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+4)]*c1*g_Vz0[index3d(iz,ix+4,it+1)]	;
		
	__syncthreads();


}


extern "C" void rtm_gpu_func(int it_max, int nt, int nz, int nx, 
        float * Vx0, float * Vz0, float * sigmaxx0, float * sigmazz0, float * sigmaxz0, //(nz, nx, nt)
        float * ex_m1_x,float * ex_m1_z,float * ex_aux_m2_c, float * ex_aux_m3_c, float * ex_aux_m2m3_c,//)//(nz+10,nx+10)
	float * debug)
{	
     	hipError_t err;
	hipEvent_t start, stop;
	float elapsedTime = 0.0f;
	int g_it;

	//time record

	//data copy in 
     	rtm_gpu_copy_in(nt, nz, nx, Vx0, Vz0, sigmaxx0, sigmazz0, sigmaxz0, ex_m1_x, ex_m1_z, ex_aux_m2_c, ex_aux_m3_c, ex_aux_m2m3_c);
	
	err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error1: %s.\n", hipGetErrorString(err));
	}	
	
	//RTM computing
	fprintf(stderr,"GPU Computing from TS=%d ... ...(NZ=%d, NX=%d, TZ=%d, TX=%d)\n", it_max, nz, nx, TZ, TX);
	for(g_it = it_max; g_it>=0; g_it--){

	dim3 dimGrid(nz/TZ, nx/TX);
	dim3 dimBlock(TZ, TX);

	rtm_gpu_kernel<<<dimGrid, dimBlock>>>(g_it,nt, nz, nx, g_Vx0, g_Vz0, g_sigmaxx0, g_sigmazz0, g_sigmaxz0, g_ex_m1_x, g_ex_m1_z, g_ex_aux_m2_c, g_ex_aux_m3_c, g_ex_aux_m2m3_c);
	hipDeviceSynchronize();

	err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error2: %s.\n", hipGetErrorString(err));
		}
	}

	//data copy out
	rtm_gpu_copy_out(nt, nz, nx, Vx0, Vz0, sigmaxx0, sigmazz0, sigmaxz0);	

	err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error3: %s.\n", hipGetErrorString(err));
	}	


	//hipEventRecord(stop, 0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&elapsedTime, start, stop);
	//fprintf(stderr,"GPU Computational Elapsed Time: %.4f\n",elapsedTime);
}

