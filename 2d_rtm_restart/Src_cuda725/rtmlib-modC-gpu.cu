#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gpu.h"
__device__ float *g_Vx0;
__device__ float *g_Vz0;
__device__ float *g_sigmaxx0; 
__device__ float *g_sigmazz0;
__device__ float *g_sigmaxz0;
__device__ float *g_m1_x;
__device__ float *g_m1_z;
__device__ float *g_aux_m2_c; 
__device__ float *g_aux_m3_c; 
__device__ float *g_aux_m2m3_c; 


//void setup_cuda(int ngpus, int argc, char **argv){
	//insert from Bob' Born
//	;
//}
//void process_error( const hipError_t &error, char *string=0, bool verbose=false ){
	//insert from Bob's Born
//	;
//}
extern "C" void rtm_gpu_init(int nt, int nz, int nx, 
        float * Vx0, float * Vz0, float * sigmaxx0, float * sigmazz0, float * sigmaxz0, //(nz, nx, nt)
        float * m1_x,float * m1_z,float * aux_m2_c, float * aux_m3_c, float * aux_m2m3_c)//(nz,	nx)
{
	//set cuda devices and put all data onto gpu memory
	
	hipError_t cuda_ret;
     	hipError_t err;

	//Set Device 
	fprintf(stderr,"GPU init. \n");
    	cuda_ret = hipSetDevice(0);
	if(cuda_ret != hipSuccess){
		fprintf(stderr, "Failed to Set The cuda Device !\n");
	}
	else{
		fprintf(stderr, "GPU Device Set OK\n");
	}

	// data init
	hipMalloc(&g_Vx0,sizeof(float)*nx*nz*nt);
	hipMalloc(&g_Vz0,sizeof(float)*nx*nz*nt);
	hipMalloc(&g_sigmaxx0,sizeof(float)*nx*nz*nt);
	hipMalloc(&g_sigmazz0,sizeof(float)*nx*nz*nt);
	hipMalloc(&g_sigmaxz0,sizeof(float)*nx*nz*nt);
	hipMalloc(&g_m1_x,sizeof(float)*nx*nz);
	hipMalloc(&g_m1_z,sizeof(float)*nx*nz);
	hipMalloc(&g_aux_m2_c,sizeof(float)*nx*nz);
	hipMalloc(&g_aux_m3_c,sizeof(float)*nx*nz);
	hipMalloc(&g_aux_m2m3_c,sizeof(float)*nx*nz);
	fprintf(stderr,"GPU Data Init OK\n");

	// data copy

//	hipMemcpy(g_Vx0, Vx0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
//	hipMemcpy(g_Vz0, Vz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
//	hipMemcpy(g_sigmaxx0, sigmaxx0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
//	hipMemcpy(g_sigmaxz0, sigmaxz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
//	hipMemcpy(g_sigmazz0, sigmazz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
//	hipMemcpy(g_m1_x, m1_x, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
//	hipMemcpy(g_m1_z, m1_z, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
//	hipMemcpy(g_aux_m2_c, aux_m2_c, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
//	hipMemcpy(g_aux_m3_c, aux_m3_c, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
//	hipMemcpy(g_aux_m2m3_c, aux_m2m3_c, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
//	fprintf(stderr,"Data Copy To GPU OK\n");
}



extern "C" void rtm_gpu_copy_in(int nt, int nz, int nx, 
        float * Vx0, float * Vz0, float * sigmaxx0, float * sigmazz0, float * sigmaxz0, //(nz, nx, nt)
        float * m1_x,float * m1_z,float * aux_m2_c, float * aux_m3_c, float * aux_m2m3_c)//(nz,	nx)
{
	// data copy

	hipMemcpy(g_Vx0, Vx0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
	hipMemcpy(g_Vz0, Vz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
	hipMemcpy(g_sigmaxx0, sigmaxx0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
	hipMemcpy(g_sigmaxz0, sigmaxz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
	hipMemcpy(g_sigmazz0, sigmazz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
	hipMemcpy(g_m1_x, m1_x, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
	hipMemcpy(g_m1_z, m1_z, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
	hipMemcpy(g_aux_m2_c, aux_m2_c, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
	hipMemcpy(g_aux_m3_c, aux_m3_c, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
	hipMemcpy(g_aux_m2m3_c, aux_m2m3_c, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
	fprintf(stderr,"Data Copy To GPU OK\n");
}


extern "C" void rtm_gpu_copy_out(int nt, int nz, int nx, 
        float * Vx0, float * Vz0, float * sigmaxx0, float * sigmazz0, float * sigmaxz0, //(nz, nx, nt)
        float * m1_x,float * m1_z,float * aux_m2_c, float * aux_m3_c, float * aux_m2m3_c)//(nz,	nx)
{
	// data copy back from GPU mem
	hipMemcpy(Vx0, g_Vx0, sizeof(float)*nx*nz*nt,  		hipMemcpyDeviceToHost);
	hipMemcpy( Vz0, g_Vz0,sizeof(float)*nx*nz*nt, 			hipMemcpyDeviceToHost);
	hipMemcpy(sigmaxx0, g_sigmaxx0, sizeof(float)*nx*nz*nt, 	hipMemcpyDeviceToHost);
	hipMemcpy(sigmaxz0, g_sigmaxz0, sizeof(float)*nx*nz*nt, 	hipMemcpyDeviceToHost);
	hipMemcpy(sigmazz0, g_sigmazz0,  sizeof(float)*nx*nz*nt, 	hipMemcpyDeviceToHost);
	hipMemcpy(m1_x, g_m1_x,  sizeof(float)*nx*nz, 			hipMemcpyDeviceToHost);
	hipMemcpy(m1_z, g_m1_z,  sizeof(float)*nx*nz, 			hipMemcpyDeviceToHost);
	hipMemcpy(aux_m2_c, g_aux_m2_c,  sizeof(float)*nx*nz, 		hipMemcpyDeviceToHost);
	hipMemcpy(aux_m3_c, g_aux_m3_c,  sizeof(float)*nx*nz, 		hipMemcpyDeviceToHost);
	hipMemcpy(aux_m2m3_c, g_aux_m2m3_c,  sizeof(float)*nx*nz, 	hipMemcpyDeviceToHost);
	fprintf(stderr,"Data Copy To CPU OK\n");

}


extern "C" void rtm_gpu_final(int nt, int nz, int nx, 
        float * Vx0, float * Vz0, float * sigmaxx0, float * sigmazz0, float * sigmaxz0, //(nz, nx, nt)
        float * m1_x,float * m1_z,float * aux_m2_c, float * aux_m3_c, float * aux_m2m3_c)//(nz,	nx)
{
	// data copy back from GPU mem
//	hipMemcpy(Vx0, g_Vx0, sizeof(float)*nx*nz*nt,  		hipMemcpyDeviceToHost);
//	hipMemcpy( Vz0, g_Vz0,sizeof(float)*nx*nz*nt, 			hipMemcpyDeviceToHost);
//	hipMemcpy(sigmaxx0, g_sigmaxx0, sizeof(float)*nx*nz*nt, 	hipMemcpyDeviceToHost);
//	hipMemcpy(sigmaxz0, g_sigmaxz0, sizeof(float)*nx*nz*nt, 	hipMemcpyDeviceToHost);
//	hipMemcpy(sigmazz0, g_sigmazz0,  sizeof(float)*nx*nz*nt, 	hipMemcpyDeviceToHost);
//	hipMemcpy(m1_x, g_m1_x,  sizeof(float)*nx*nz, 			hipMemcpyDeviceToHost);
//	hipMemcpy(m1_z, g_m1_z,  sizeof(float)*nx*nz, 			hipMemcpyDeviceToHost);
//	hipMemcpy(aux_m2_c, g_aux_m2_c,  sizeof(float)*nx*nz, 		hipMemcpyDeviceToHost);
//	hipMemcpy(aux_m3_c, g_aux_m3_c,  sizeof(float)*nx*nz, 		hipMemcpyDeviceToHost);
//	hipMemcpy(aux_m2m3_c, g_aux_m2m3_c,  sizeof(float)*nx*nz, 	hipMemcpyDeviceToHost);
//	fprintf(stderr,"Data Copy To CPU OK\n");


	hipFree(&g_Vx0);
	hipFree(&g_Vz0);
	hipFree(&g_sigmaxx0);
	hipFree(&g_sigmazz0);
	hipFree(&g_sigmaxz0);
	hipFree(&g_m1_x);
	hipFree(&g_m1_z);
	hipFree(&g_aux_m2_c);
	hipFree(&g_aux_m3_c);
	hipFree(&g_aux_m2m3_c);
	fprintf(stderr,"GPU Mem Released OK\n");
}


__global__ void rtm_gpu_kernel(int it,int nt, int nz, int nx, 
        float * Vx0, float * Vz0, float * sigmaxx0, float * sigmazz0, float * sigmaxz0) //(nz, nx, nt)
        //float * m1_x,float * m1_z,float * aux_m2_c, float * aux_m3_c, float * aux_m2m3_c)//(nz,	nx)
{
	//GPU thread index
	int gz, gx, gt;
	gz = blockIdx.x*blockDim.x + threadIdx.x;
	gx = blockIdx.y*blockDim.y + threadIdx.y;
	gt = it;
//	gt = blockIdx.z*blockDim.y + threadIdx.z;
 
       Vx0[index3d(gz, gx, gt)] = Vx0[index3d(gz, gx, gt)] + Vx0[index3d(gz, gx, gt+2)];
       Vz0[index3d(gz, gx, gt)] = Vz0[index3d(gz, gx, gt)] + Vz0[index3d(gz, gx, gt+2)];
       sigmaxx0[index3d(gz, gx, gt)] = sigmaxx0[index3d(gz, gx, gt)] + sigmaxx0[index3d(gz, gx, gt+2)];
       sigmazz0[index3d(gz, gx, gt)] = sigmazz0[index3d(gz, gx, gt)] + sigmazz0[index3d(gz, gx, gt+2)];
       sigmaxz0[index3d(gz, gx, gt)] = sigmaxz0[index3d(gz, gx, gt)] + sigmaxz0[index3d(gz, gx, gt+2)];
}


extern "C" void rtm_gpu_func(int it, int nt, int nz, int nx, 
        float * Vx0, float * Vz0, float * sigmaxx0, float * sigmazz0, float * sigmaxz0, //(nz, nx, nt)
        float * m1_x,float * m1_z,float * aux_m2_c, float * aux_m3_c, float * aux_m2m3_c)//(nz,	nx)
{	
     	hipError_t err;
	hipEvent_t start, stop;
	float elapsedTime = 0.0f;

	//time record
	
	dim3 dimGrid(nz/TZ, nx/TX);
	dim3 dimBlock(TZ, TX);

	//RTM kernel 
	fprintf(stderr,"GPU Computing...(NZ=%d, NX=%d, TZ=%d, TX=%d)\n", nz, nx, TZ, TX);

	//hipEventRecord(start, 0);
	rtm_gpu_kernel<<<dimGrid, dimBlock>>>(it,nt, nz, nx, g_Vx0, g_Vz0, g_sigmaxx0, g_sigmazz0, g_sigmaxz0);
	hipDeviceSynchronize();

	err = hipGetLastError();
	if(hipSuccess != err)
		fprintf(stderr, "Cuda error: %s.\n", hipGetErrorString(err));


	//hipEventRecord(stop, 0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&elapsedTime, start, stop);
	//fprintf(stderr,"GPU Computational Elapsed Time: %.4f\n",elapsedTime);



	
}

