
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float *g_Vx0;
__device__ float *g_Vz0;
__device__ float *g_sigmaxx0; 
__device__ float *g_sigmazz0;
__device__ float *g_sigmaxz0;
__device__ float *g_m1_x;
__device__ float *g_m1_z;
__device__ float *g_aux_m2_c; 
__device__ float *g_aux_m3_c; 
__device__ float *g_aux_m2m3_c; 
//void setup_cuda(int ngpus, int argc, char **argv){
	//insert from Bob' Born
//	;
//}
//void process_error( const cudaError_t &error, char *string=0, bool verbose=false ){
	//insert from Bob's Born
//	;
//}


extern "C" void rtm_gpu_init(int nt, int nz, int nx, 
        float * Vx0, float * Vz0, float * sigmaxx0, float * sigmazz0, float * sigmaxz0, //(nz, nx, nt)
//        float * Vx,  float * Vz,  float * sigmaxx,  float * sigmazz,  float * sigmaxz, //(nt, nx)
        float * m1_x,float * m1_z,float * aux_m2_c, float * aux_m3_c, float * aux_m2m3_c)
{
	//set cuda devices and put all data onto gpu memory
	
	hipError_t cuda_ret;
     	hipError_t err;

	//Set Device 
	fprintf(stderr,"GPU init. \n");
    	cuda_ret = hipSetDevice(0);
	if(cuda_ret != hipSuccess){
		fprintf(stderr, "Failed to Set The cuda Device !\n");
	}
	else{
		fprintf(stderr, "GPU Device Set OK\n");
	}

	// data init
	hipMalloc(&g_Vx0,sizeof(float)*nx*nz*nt);
	hipMalloc(&g_Vz0,sizeof(float)*nx*nz*nt);
	hipMalloc(&g_sigmaxx0,sizeof(float)*nx*nz*nt);
	hipMalloc(&g_sigmazz0,sizeof(float)*nx*nz*nt);
	hipMalloc(&g_sigmaxz0,sizeof(float)*nx*nz*nt);
	hipMalloc(&g_m1_x,sizeof(float)*nx*nz);
	hipMalloc(&g_m1_z,sizeof(float)*nx*nz);
	hipMalloc(&g_aux_m2_c,sizeof(float)*nx*nz);
	hipMalloc(&g_aux_m3_c,sizeof(float)*nx*nz);
	hipMalloc(&g_aux_m2m3_c,sizeof(float)*nx*nz);
	fprintf(stderr,"GPU Data Init OK\n");

	// data copy

	hipMemcpy(g_Vx0, Vx0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
	hipMemcpy(g_Vz0, Vz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
	hipMemcpy(g_sigmaxx0, sigmaxx0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
	hipMemcpy(g_sigmaxz0, sigmaxz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
	hipMemcpy(g_sigmazz0, sigmazz0, sizeof(float)*nx*nz*nt, hipMemcpyHostToDevice);
	hipMemcpy(g_m1_x, m1_x, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
	hipMemcpy(g_m1_z, m1_z, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
	hipMemcpy(g_aux_m2_c, aux_m2_c, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
	hipMemcpy(g_aux_m3_c, aux_m3_c, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
	hipMemcpy(g_aux_m2m3_c, aux_m2m3_c, sizeof(float)*nx*nz, hipMemcpyHostToDevice);
	fprintf(stderr,"Data Copy To GPU OK\n");

	
	// data copy back from GPU mem
	hipMemcpy(Vx0, g_Vx0, sizeof(float)*nx*nz*nt,  		hipMemcpyDeviceToHost);
	hipMemcpy( Vz0, g_Vz0,sizeof(float)*nx*nz*nt, 			hipMemcpyDeviceToHost);
	hipMemcpy(sigmaxx0, g_sigmaxx0, sizeof(float)*nx*nz*nt, 		hipMemcpyDeviceToHost);
	hipMemcpy(sigmaxz0, g_sigmaxz0, sizeof(float)*nx*nz*nt, 		hipMemcpyDeviceToHost);
	hipMemcpy(sigmazz0, g_sigmazz0,  sizeof(float)*nx*nz*nt, 	hipMemcpyDeviceToHost);
	hipMemcpy(m1_x, g_m1_x,  sizeof(float)*nx*nz, 			hipMemcpyDeviceToHost);
	hipMemcpy(m1_z, g_m1_z,  sizeof(float)*nx*nz, 			hipMemcpyDeviceToHost);
	hipMemcpy(aux_m2_c, g_aux_m2_c,  sizeof(float)*nx*nz, 		hipMemcpyDeviceToHost);
	hipMemcpy(aux_m3_c, g_aux_m3_c,  sizeof(float)*nx*nz, 		hipMemcpyDeviceToHost);
	hipMemcpy(aux_m2m3_c, g_aux_m2m3_c,  sizeof(float)*nx*nz, 	hipMemcpyDeviceToHost);
	fprintf(stderr,"Data Copy To CPU OK\n");


	hipFree(&g_Vx0);
	hipFree(&g_Vz0);
	hipFree(&g_sigmaxx0);
	hipFree(&g_sigmazz0);
	hipFree(&g_sigmaxz0);
	hipFree(&g_m1_x);
	hipFree(&g_m1_z);
	hipFree(&g_aux_m2_c);
	hipFree(&g_aux_m3_c);
	hipFree(&g_aux_m2m3_c);
	fprintf(stderr,"GPU Mem Released OK\n");
	

}

