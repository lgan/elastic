#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gpu.h"



//extended propagation data residence in GPU device

__device__	float * g_ex_Vx0_in;
__device__	float * g_ex_Vz0_in;
__device__	float * g_ex_Vy0_in;
__device__	float * g_ex_sigmaxx0_in;
__device__	float * g_ex_sigmazz0_in;
__device__	float * g_ex_sigmayy0_in;
__device__	float * g_ex_sigmaxy0_in;
__device__	float * g_ex_sigmaxz0_in;
__device__	float * g_ex_sigmayz0_in;
	
		//Time step +2
__device__	float * g_ex_Vx0_in1;
__device__	float * g_ex_Vz0_in1;
__device__	float * g_ex_Vy0_in1;
__device__	float * g_ex_sigmaxx0_in1;
__device__	float * g_ex_sigmazz0_in1;
__device__	float * g_ex_sigmayy0_in1;
__device__	float * g_ex_sigmaxy0_in1;
__device__	float * g_ex_sigmaxz0_in1;
__device__	float * g_ex_sigmayz0_in1;


		//time step 0 and output
__device__	float * g_ex_Vx0_out;
__device__	float * g_ex_Vz0_out;
__device__	float * g_ex_Vy0_out;
__device__	float * g_ex_sigmaxx0_out;
__device__	float * g_ex_sigmazz0_out;
__device__	float * g_ex_sigmayy0_out;
__device__	float * g_ex_sigmaxy0_out;
__device__	float * g_ex_sigmaxz0_out;
__device__	float * g_ex_sigmayz0_out;

	
 
   
		//expaned arrays to store different Operators 
__device__	float *g_ex_m2;
__device__	float *g_ex_m3;
__device__	float *g_ex_m2m3;
__device__	float *g_ex_m1_x;
__device__	float *g_ex_m1_z;
__device__	float *g_ex_m1_y;


__device__	float *g_tmp;



__global__ void rtm_gpu_kernel(int ny, int nz, int nx,
        float *g_ex_Vy0_in,  float * g_ex_Vx0_in, float * g_ex_Vz0_in, float * g_ex_sigmayy0_in, float *g_ex_sigmaxx0_in, float * g_ex_sigmazz0_in, float * g_ex_sigmaxy0_in, float * g_ex_sigmaxz0_in, float * g_ex_sigmayz0_in,//(nz, nx, nt)
        float *g_ex_Vy0_in1,  float * g_ex_Vx0_in1, float * g_ex_Vz0_in1, float * g_ex_sigmayy0_in1, float *g_ex_sigmaxx0_in1, float * g_ex_sigmazz0_in1, float * g_ex_sigmaxy0_in1, float * g_ex_sigmaxz0_in1, float * g_ex_sigmayz0_in1,//(nz, nx, nt)
        float *g_ex_Vy0_out,  float * g_ex_Vx0_out, float * g_ex_Vz0_out, float * g_ex_sigmayy0_out, float *g_ex_sigmaxx0_out, float * g_ex_sigmazz0_out, float * g_ex_sigmaxy0_out, float * g_ex_sigmaxz0_out, float * g_ex_sigmayz0_out,//(nz, nx, nt)
     	const float * __restrict__ g_ex_m1_y, 	const float * __restrict__ g_ex_m1_x,	const float * __restrict__ g_ex_m1_z, const float * __restrict__  g_ex_m2, const float * __restrict__  g_ex_m3, const float * __restrict__  g_ex_m2m3);//(nz+10,	nx+10)



extern "C" void rtm_gpu_init(int ny, int nz, int nx) 
{
	//set cuda devices and put all data onto gpu memory
	
	hipError_t cuda_ret;
     	hipError_t err;

	//Set Device 
    	cuda_ret = hipSetDevice(0);
	if(cuda_ret != hipSuccess){
		fprintf(stderr, "Failed to Set The cuda Device !\n");
	}
	else{
		fprintf(stderr, "GPU Device Set ====> OK\n");
	}

	// data init
	
	//Time step +1
	hipMalloc(&g_ex_Vx0_in, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_Vz0_in, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_Vy0_in, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxx0_in, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmazz0_in, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmayy0_in, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxy0_in, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxz0_in, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmayz0_in, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	
	//Time step +2
	hipMalloc(&g_ex_Vx0_in1, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_Vz0_in1, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_Vy0_in1, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxx0_in1, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmazz0_in1, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmayy0_in1, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxy0_in1, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxz0_in1, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmayz0_in1, sizeof(float)*(ny+10)*(nx+10)*(nz+10));


	//time step 0 and output
	hipMalloc(&g_ex_Vx0_out, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_Vz0_out, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_Vy0_out, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxx0_out, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmazz0_out, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmayy0_out, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxy0_out, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmaxz0_out, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_sigmayz0_out, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
   
	//expaned arrays to store different Operators 
	hipMalloc(&g_ex_m2, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_m3, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_m2m3, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_m1_x, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_m1_y, sizeof(float)*(ny+10)*(nx+10)*(nz+10));
	hipMalloc(&g_ex_m1_z, sizeof(float)*(ny+10)*(nx+10)*(nz+10));

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(rtm_gpu_kernel),hipFuncCachePreferShared);
	
	err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error1: %s.\n", hipGetErrorString(err));
	}else{	
		fprintf(stderr,"GPU Data Init ====> OK\n");
	}
	// data copy

}



extern "C" void rtm_gpu_copy_in(int ny, int nz, int nx, 
        float *ex_Vy0_in,  float * ex_Vx0_in, float * ex_Vz0_in, float * ex_sigmayy0_in, float *ex_sigmaxx0_in, float * ex_sigmazz0_in, float * ex_sigmaxy0_in, float * ex_sigmaxz0_in, float * ex_sigmayz0_in,//(nz, nx, nt)
        float *ex_Vy0_in1,  float * ex_Vx0_in1, float * ex_Vz0_in1, float * ex_sigmayy0_in1, float *ex_sigmaxx0_in1, float * ex_sigmazz0_in1, float * ex_sigmaxy0_in1, float * ex_sigmaxz0_in1, float * ex_sigmayz0_in1,//(nz, nx, nt)
        float *ex_Vy0_out,  float * ex_Vx0_out, float * ex_Vz0_out, float * ex_sigmayy0_out, float *ex_sigmaxx0_out, float * ex_sigmazz0_out, float * ex_sigmaxy0_out, float * ex_sigmaxz0_out, float * ex_sigmayz0_out,//(nz, nx, nt)
     	float * ex_m1_y,  float * ex_m1_x, float * ex_m1_z, float * ex_m2, float * ex_m3, float * ex_m2m3)//(nz+10,	nx+10)
  {
	// data copy

	hipMemcpy(g_ex_Vy0_in, ex_Vy0_in, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_Vx0_in, ex_Vx0_in, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_Vz0_in, ex_Vz0_in, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxx0_in, ex_sigmaxx0_in, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmayy0_in, ex_sigmayy0_in, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxy0_in, ex_sigmaxy0_in, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmayz0_in, ex_sigmayz0_in, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxz0_in, ex_sigmaxz0_in, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmazz0_in, ex_sigmazz0_in, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);

	hipMemcpy(g_ex_Vy0_in1, ex_Vy0_in1, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_Vx0_in1, ex_Vx0_in1, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_Vz0_in1, ex_Vz0_in1, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxx0_in1, ex_sigmaxx0_in1, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmayy0_in1, ex_sigmayy0_in1, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxy0_in1, ex_sigmaxy0_in1, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmayz0_in1, ex_sigmayz0_in1, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxz0_in1, ex_sigmaxz0_in1, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmazz0_in1, ex_sigmazz0_in1, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);

	hipMemcpy(g_ex_Vy0_out, ex_Vy0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_Vx0_out, ex_Vx0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_Vz0_out, ex_Vz0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxx0_out, ex_sigmaxx0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmayy0_out, ex_sigmayy0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxy0_out, ex_sigmaxy0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmayz0_out, ex_sigmayz0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmaxz0_out, ex_sigmaxz0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_sigmazz0_out, ex_sigmazz0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), hipMemcpyHostToDevice);



	hipMemcpy(g_ex_m1_y, ex_m1_y, sizeof(float)*(ny+10)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_m1_x, ex_m1_x, sizeof(float)*(ny+10)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_m1_z, ex_m1_z, sizeof(float)*(ny+10)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_m2, ex_m2, sizeof(float)*(ny+10)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_m3, ex_m3, sizeof(float)*(ny+10)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	hipMemcpy(g_ex_m2m3, ex_m2m3, sizeof(float)*(ny+10)*(nx+10)*(nz+10), hipMemcpyHostToDevice);
	
	err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error1: %s.\n", hipGetErrorString(err));
	}else{
		fprintf(stderr,"Data Copy To GPU  ====> OK\n");
	}
}



extern "C" void rtm_gpu_copy_out(int ny, int nz, int nx, 
        float *ex_Vy0_out,  float * ex_Vx0_out, float * ex_Vz0_out, float * ex_sigmayy0_out, float *ex_sigmaxx0_out, float * ex_sigmazz0_out, float * ex_sigmaxy0_out, float * ex_sigmaxz0_out, float * ex_sigmayz0_out)//(nz, nx, nt)
{
	// data copy back from GPU mem
	hipMemcpy(ex_Vy0_out, g_ex_Vy0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10),  		hipMemcpyDeviceToHost);
	hipMemcpy(ex_Vx0_out, g_ex_Vx0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10),  		hipMemcpyDeviceToHost);
	hipMemcpy(ex_Vz0_out, g_ex_Vz0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), 			hipMemcpyDeviceToHost);
	hipMemcpy(ex_sigmaxx0_out, g_ex_sigmaxx0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), 	hipMemcpyDeviceToHost);
	hipMemcpy(ex_sigmayy0_out, g_ex_sigmayy0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), 	hipMemcpyDeviceToHost);
	hipMemcpy(ex_sigmaxy0_out, g_ex_sigmaxy0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), 	hipMemcpyDeviceToHost);
	hipMemcpy(ex_sigmaxz0_out, g_ex_sigmaxz0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), 	hipMemcpyDeviceToHost);
	hipMemcpy(ex_sigmayz0_out, g_ex_sigmayz0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), 	hipMemcpyDeviceToHost);
	hipMemcpy(ex_sigmazz0_out, g_ex_sigmazz0_out, sizeof(float)*(nx+10)*(nz+10)*(ny+10), 	hipMemcpyDeviceToHost);
	//hipMemcpy(sigmazz0, g_sigmazz0,  sizeof(float)*nx*nz*nt, 	hipMemcpyDeviceToHost);
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error1: %s.\n", hipGetErrorString(err));
	}else{
	fprintf(stderr,"Data Copy To CPU ====> OK\n");
	}
}


extern "C" void rtm_gpu_final()
{

	//release GPU memory space


	hipFree(&g_ex_Vx0_in);
	hipFree(&g_ex_Vz0_in);
	hipFree(&g_ex_Vy0_in);
	hipFree(&g_ex_sigmaxx0_in);
	hipFree(&g_ex_sigmazz0_in);
	hipFree(&g_ex_sigmayy0_in);
	hipFree(&g_ex_sigmaxy0_in);
	hipFree(&g_ex_sigmaxz0_in);
	hipFree(&g_ex_sigmayz0_in);
	
	//Time step +2
	hipFree(&g_ex_Vx0_in1);
	hipFree(&g_ex_Vz0_in1);
	hipFree(&g_ex_Vy0_in1);
	hipFree(&g_ex_sigmaxx0_in1);
	hipFree(&g_ex_sigmazz0_in1);
	hipFree(&g_ex_sigmayy0_in1);
	hipFree(&g_ex_sigmaxy0_in1);
	hipFree(&g_ex_sigmaxz0_in1);
	hipFree(&g_ex_sigmayz0_in1);


	//time step 0 and output
	hipFree(&g_ex_Vx0_out);
	hipFree(&g_ex_Vz0_out);
	hipFree(&g_ex_Vy0_out);
	hipFree(&g_ex_sigmaxx0_out);
	hipFree(&g_ex_sigmazz0_out);
	hipFree(&g_ex_sigmayy0_out);
	hipFree(&g_ex_sigmaxy0_out);
	hipFree(&g_ex_sigmaxz0_out);
	hipFree(&g_ex_sigmayz0_out);
   
	//expaned arrays to store different Operators 
	hipFree(&g_ex_m2);
	hipFree(&g_ex_m3);
	hipFree(&g_ex_m2m3);
	hipFree(&g_ex_m1_x);
	hipFree(&g_ex_m1_y);
	hipFree(&g_ex_m1_z);

	if(hipSuccess != err){
		fprintf(stderr, "Cuda error1: %s.\n", hipGetErrorString(err));
	}else{
	fprintf(stderr,"GPU Mem Released ====> OK\n");
	}
}


void rtm_gpu_change_pointer(){
		
			
		fprintf(stderr, "GPU pointer changed\n");

		g_tmp = 	g_ex_Vx0_out;
		g_ex_Vx0_out = 	g_ex_Vx0_in;
		g_ex_Vx0_in = 	g_tmp;

		g_tmp = 	g_ex_Vx0_out;
		g_ex_Vx0_out = 	g_ex_Vx0_in1;
		g_ex_Vx0_in1 = 	g_tmp; 

	

		g_tmp = 	g_ex_Vz0_out;
		g_ex_Vz0_out = 	g_ex_Vz0_in;
		g_ex_Vz0_in = 	g_tmp;

		g_tmp = 	g_ex_Vz0_out;
		g_ex_Vz0_out = 	g_ex_Vz0_in1;
		g_ex_Vz0_in1 = 	g_tmp; 


	
		g_tmp = 	g_ex_Vy0_out;
		g_ex_Vy0_out = 	g_ex_Vy0_in;
		g_ex_Vy0_in = 	g_tmp;

		g_tmp = 	g_ex_Vy0_out;
		g_ex_Vy0_out = 	g_ex_Vy0_in1;
		g_ex_Vy0_in1 = 	g_tmp; 

	

		g_tmp = 	g_ex_sigmaxx0_out;
		g_ex_sigmaxx0_out = g_ex_sigmaxx0_in;
		g_ex_sigmaxx0_in = g_tmp;

		g_tmp = g_ex_sigmaxx0_out;
		g_ex_sigmaxx0_out = g_ex_sigmaxx0_in1;
		g_ex_sigmaxx0_in1 = g_tmp; 



	
		g_tmp = g_ex_sigmazz0_out;
		g_ex_sigmazz0_out = g_ex_sigmazz0_in;
		g_ex_sigmazz0_in = g_tmp;

		g_tmp = g_ex_sigmazz0_out;
		g_ex_sigmazz0_out = g_ex_sigmazz0_in1;
		g_ex_sigmazz0_in1 = g_tmp; 



	
		g_tmp = g_ex_sigmayy0_out;
		g_ex_sigmayy0_out = g_ex_sigmayy0_in;
		g_ex_sigmayy0_in = g_tmp;

		g_tmp = g_ex_sigmayy0_out;
		g_ex_sigmayy0_out = g_ex_sigmayy0_in1;
		g_ex_sigmayy0_in1 = g_tmp; 


	
		g_tmp = g_ex_sigmaxy0_out;
		g_ex_sigmaxy0_out = g_ex_sigmaxy0_in;
		g_ex_sigmaxy0_in = g_tmp;

		g_tmp = g_ex_sigmaxy0_out;
		g_ex_sigmaxy0_out = g_ex_sigmaxy0_in1;
		g_ex_sigmaxy0_in1 = g_tmp; 



	
		g_tmp = g_ex_sigmaxz0_out;
		g_ex_sigmaxz0_out = g_ex_sigmaxz0_in;
		g_ex_sigmaxz0_in = g_tmp;

		g_tmp = g_ex_sigmaxz0_out;
		g_ex_sigmaxz0_out = g_ex_sigmaxz0_in1;
		g_ex_sigmaxz0_in1 = g_tmp; 


	
		g_tmp = g_ex_sigmayz0_out;
		g_ex_sigmayz0_out = g_ex_sigmayz0_in;
		g_ex_sigmayz0_in = g_tmp;
		
		g_tmp = g_ex_sigmayz0_out;
		g_ex_sigmayz0_out = g_ex_sigmayz0_in1;
		g_ex_sigmayz0_in1 = g_tmp; 



}


__global__ void rtm_gpu_kernel(int ny, int nz, int nx,
        float *g_ex_Vy0_in,  float * g_ex_Vx0_in, float * g_ex_Vz0_in, float * g_ex_sigmayy0_in, float *g_ex_sigmaxx0_in, float * g_ex_sigmazz0_in, float * g_ex_sigmaxy0_in, float * g_ex_sigmaxz0_in, float * g_ex_sigmayz0_in,//(nz, nx, nt)
        float *g_ex_Vy0_in1,  float * g_ex_Vx0_in1, float * g_ex_Vz0_in1, float * g_ex_sigmayy0_in1, float *g_ex_sigmaxx0_in1, float * g_ex_sigmazz0_in1, float * g_ex_sigmaxy0_in1, float * g_ex_sigmaxz0_in1, float * g_ex_sigmayz0_in1,//(nz, nx, nt)
        float *g_ex_Vy0_out,  float * g_ex_Vx0_out, float * g_ex_Vz0_out, float * g_ex_sigmayy0_out, float *g_ex_sigmaxx0_out, float * g_ex_sigmazz0_out, float * g_ex_sigmaxy0_out, float * g_ex_sigmaxz0_out, float * g_ex_sigmayz0_out,//(nz, nx, nt)
     	const float * __restrict__ g_ex_m1_y, 	const float * __restrict__ g_ex_m1_x,	const float * __restrict__ g_ex_m1_z, const float * __restrict__  g_ex_m2, const float * __restrict__  g_ex_m3, const float * __restrict__  g_ex_m2m3)//(nz+10,	nx+10)
{

	float c1=35.0/294912.0,c2=-405.0/229376.0,c3=567.0/40960.0,c4=-735.0/8192.0,c5=19845.0/16384.0;

	//GPU thread index
	int iz, ix, iy;
	iz = blockIdx.x*blockDim.x + threadIdx.x;
	ix = blockIdx.y*blockDim.y + threadIdx.y;
	iy = blockIdx.z*blockDim.z + threadIdx.z;
	//gt = it;
  
       	g_ex_Vx0_out[n3d_index_ex(iz,ix  ,iy)] = g_ex_Vx0_out[n3d_index_ex(iz,ix  ,iy)]	+ g_ex_Vx0_in1[n3d_index_ex(iz, ix, iy)]	

									+ g_ex_m2m3[n3d_index_ex(iz,ix-5, iy)]*c1*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix-5,iy)]							
							 		+ g_ex_m2m3[n3d_index_ex(iz,ix-4, iy)]*c2*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m2m3[n3d_index_ex(iz,ix-3, iy)]*c3*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m2m3[n3d_index_ex(iz,ix-2, iy)]*c4*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m2m3[n3d_index_ex(iz,ix-1, iy)]*c5*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix-1,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix+1, iy)]*c4*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix+2, iy)]*c3*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix+3, iy)]*c2*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix+4, iy)]*c1*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix+4,iy)]
	

									+ g_ex_m2[n3d_index_ex(iz,ix-5, iy)]*c1*g_ex_sigmayy0_in[n3d_index_ex(iz,ix-5,iy)]							
							 		+ g_ex_m2[n3d_index_ex(iz,ix-4, iy)]*c2*g_ex_sigmayy0_in[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m2[n3d_index_ex(iz,ix-3, iy)]*c3*g_ex_sigmayy0_in[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m2[n3d_index_ex(iz,ix-2, iy)]*c4*g_ex_sigmayy0_in[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m2[n3d_index_ex(iz,ix-1, iy)]*c5*g_ex_sigmayy0_in[n3d_index_ex(iz,ix-1,iy)]	
									- g_ex_m2[n3d_index_ex(iz,  ix, iy)]*c5*g_ex_sigmayy0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+1, iy)]*c4*g_ex_sigmayy0_in[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+2, iy)]*c3*g_ex_sigmayy0_in[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+3, iy)]*c2*g_ex_sigmayy0_in[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+4, iy)]*c1*g_ex_sigmayy0_in[n3d_index_ex(iz,ix+4,iy)]	
	

									+ g_ex_m2[n3d_index_ex(iz,ix-5, iy)]*c1*g_ex_sigmazz0_in[n3d_index_ex(iz,ix-5,iy)]							
							 		+ g_ex_m2[n3d_index_ex(iz,ix-4, iy)]*c2*g_ex_sigmazz0_in[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m2[n3d_index_ex(iz,ix-3, iy)]*c3*g_ex_sigmazz0_in[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m2[n3d_index_ex(iz,ix-2, iy)]*c4*g_ex_sigmazz0_in[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m2[n3d_index_ex(iz,ix-1, iy)]*c5*g_ex_sigmazz0_in[n3d_index_ex(iz,ix-1,iy)]	
									- g_ex_m2[n3d_index_ex(iz,  ix, iy)]*c5*g_ex_sigmazz0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+1, iy)]*c4*g_ex_sigmazz0_in[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+2, iy)]*c3*g_ex_sigmazz0_in[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+3, iy)]*c2*g_ex_sigmazz0_in[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix+4, iy)]*c1*g_ex_sigmazz0_in[n3d_index_ex(iz,ix+4,iy)]	
	

									+ g_ex_m3[n3d_index_ex(iz,ix, iy-4)]*c1*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-3)]*c2*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-2)]*c3*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-1)]*c4*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix,iy-1)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+1)]*c5*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+2)]*c4*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+3)]*c3*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+4)]*c2*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix,iy+4)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+5)]*c1*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix,iy+5)]							
	

									+ g_ex_m3[n3d_index_ex(iz-4,ix, iy)]*c1*g_ex_sigmaxz0_in[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m3[n3d_index_ex(iz-3,ix, iy)]*c2*g_ex_sigmaxz0_in[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz-2,ix, iy)]*c3*g_ex_sigmaxz0_in[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz-1,ix, iy)]*c4*g_ex_sigmaxz0_in[n3d_index_ex(iz-1,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,  ix, iy)]*c5*g_ex_sigmaxz0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+1,ix, iy)]*c5*g_ex_sigmaxz0_in[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+2,ix, iy)]*c4*g_ex_sigmaxz0_in[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+3,ix, iy)]*c3*g_ex_sigmaxz0_in[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+4,ix, iy)]*c2*g_ex_sigmaxz0_in[n3d_index_ex(iz+4,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+5,ix, iy)]*c1*g_ex_sigmaxz0_in[n3d_index_ex(iz+5,ix,iy)]	;						
	


         	g_ex_Vy0_out[n3d_index_ex(iz,ix  ,iy)] = g_ex_Vy0_out[n3d_index_ex(iz,ix  ,iy)]	+ g_ex_Vy0_in1[n3d_index_ex(iz, ix, iy)]	

									+ g_ex_m2m3[n3d_index_ex(iz,ix, iy-5)]*c1*g_ex_sigmayy0_in[n3d_index_ex(iz,ix,iy-5)]							
							 		+ g_ex_m2m3[n3d_index_ex(iz,ix, iy-4)]*c2*g_ex_sigmayy0_in[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m2m3[n3d_index_ex(iz,ix, iy-3)]*c3*g_ex_sigmayy0_in[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m2m3[n3d_index_ex(iz,ix, iy-2)]*c4*g_ex_sigmayy0_in[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m2m3[n3d_index_ex(iz,ix, iy-1)]*c5*g_ex_sigmayy0_in[n3d_index_ex(iz,ix,iy-1)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmayy0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy+1)]*c4*g_ex_sigmayy0_in[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy+2)]*c3*g_ex_sigmayy0_in[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy+3)]*c2*g_ex_sigmayy0_in[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy+4)]*c1*g_ex_sigmayy0_in[n3d_index_ex(iz,ix,iy+4)]
	

									+ g_ex_m2[n3d_index_ex(iz,ix, iy-5)]*c1*g_ex_sigmazz0_in[n3d_index_ex(iz,ix,iy-5)]							
							 		+ g_ex_m2[n3d_index_ex(iz,ix, iy-4)]*c2*g_ex_sigmazz0_in[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-3)]*c3*g_ex_sigmazz0_in[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-2)]*c4*g_ex_sigmazz0_in[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-1)]*c5*g_ex_sigmazz0_in[n3d_index_ex(iz,ix,iy-1)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmazz0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+1)]*c4*g_ex_sigmazz0_in[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+2)]*c3*g_ex_sigmazz0_in[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+3)]*c2*g_ex_sigmazz0_in[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+4)]*c1*g_ex_sigmazz0_in[n3d_index_ex(iz,ix,iy+4)]	
	

									+ g_ex_m2[n3d_index_ex(iz,ix, iy-5)]*c1*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix,iy-5)]							
							 		+ g_ex_m2[n3d_index_ex(iz,ix, iy-4)]*c2*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-3)]*c3*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-2)]*c4*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m2[n3d_index_ex(iz,ix, iy-1)]*c5*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix,iy-1)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+1)]*c4*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+2)]*c3*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+3)]*c2*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy+4)]*c1*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix,iy+4)]	
	

									+ g_ex_m3[n3d_index_ex(iz-4,ix, iy)]*c1*g_ex_sigmayz0_in[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m3[n3d_index_ex(iz-3,ix, iy)]*c2*g_ex_sigmayz0_in[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz-2,ix, iy)]*c3*g_ex_sigmayz0_in[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz-1,ix, iy)]*c4*g_ex_sigmayz0_in[n3d_index_ex(iz-1,ix,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmayz0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+1,ix, iy)]*c5*g_ex_sigmayz0_in[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+2,ix, iy)]*c4*g_ex_sigmayz0_in[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+3,ix, iy)]*c3*g_ex_sigmayz0_in[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+4,ix, iy)]*c2*g_ex_sigmayz0_in[n3d_index_ex(iz+4,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz+5,ix, iy)]*c1*g_ex_sigmayz0_in[n3d_index_ex(iz+5,ix,iy)]							
	

									+ g_ex_m3[n3d_index_ex(iz,ix-4, iy)]*c1*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m3[n3d_index_ex(iz,ix-3, iy)]*c2*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix-2, iy)]*c3*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix-1, iy)]*c4*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix-1,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+1, iy)]*c5*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+2, iy)]*c4*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+3, iy)]*c3*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+4, iy)]*c2*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix+4,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+5, iy)]*c1*g_ex_sigmaxy0_in[n3d_index_ex(iz,ix+5,iy)]	;						




         	g_ex_Vz0_out[n3d_index_ex(iz,ix  ,iy)] = g_ex_Vz0_out[n3d_index_ex(iz,ix  ,iy)]	+ g_ex_Vz0_in1[n3d_index_ex(iz, ix, iy)]	

									+ g_ex_m2m3[n3d_index_ex(iz-5,ix, iy)]*c1*g_ex_sigmazz0_in[n3d_index_ex(iz-5,ix,iy)]							
							 		+ g_ex_m2m3[n3d_index_ex(iz-4,ix, iy)]*c2*g_ex_sigmazz0_in[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m2m3[n3d_index_ex(iz-3,ix, iy)]*c3*g_ex_sigmazz0_in[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m2m3[n3d_index_ex(iz-2,ix, iy)]*c4*g_ex_sigmazz0_in[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m2m3[n3d_index_ex(iz-1,ix, iy)]*c5*g_ex_sigmazz0_in[n3d_index_ex(iz-1,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmazz0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz+1,ix, iy)]*c4*g_ex_sigmazz0_in[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz+2,ix, iy)]*c3*g_ex_sigmazz0_in[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz+3,ix, iy)]*c2*g_ex_sigmazz0_in[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m2m3[n3d_index_ex(iz+4,ix, iy)]*c1*g_ex_sigmazz0_in[n3d_index_ex(iz+4,ix,iy)]
	

									+ g_ex_m2[n3d_index_ex(iz-5,ix, iy)]*c1*g_ex_sigmaxx0_in[n3d_index_ex(iz-5,ix,iy)]							
							 		+ g_ex_m2[n3d_index_ex(iz-4,ix, iy)]*c2*g_ex_sigmaxx0_in[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m2[n3d_index_ex(iz-3,ix, iy)]*c3*g_ex_sigmaxx0_in[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m2[n3d_index_ex(iz-2,ix, iy)]*c4*g_ex_sigmaxx0_in[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m2[n3d_index_ex(iz-1,ix, iy)]*c5*g_ex_sigmaxx0_in[n3d_index_ex(iz-1,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxx0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+1,ix, iy)]*c4*g_ex_sigmaxx0_in[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+2,ix, iy)]*c3*g_ex_sigmaxx0_in[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+3,ix, iy)]*c2*g_ex_sigmaxx0_in[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+4,ix, iy)]*c1*g_ex_sigmaxx0_in[n3d_index_ex(iz+4,ix,iy)]
	

									+ g_ex_m2[n3d_index_ex(iz-5,ix, iy)]*c1*g_ex_sigmayy0_in[n3d_index_ex(iz-5,ix,iy)]							
							 		+ g_ex_m2[n3d_index_ex(iz-4,ix, iy)]*c2*g_ex_sigmayy0_in[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m2[n3d_index_ex(iz-3,ix, iy)]*c3*g_ex_sigmayy0_in[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m2[n3d_index_ex(iz-2,ix, iy)]*c4*g_ex_sigmayy0_in[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m2[n3d_index_ex(iz-1,ix, iy)]*c5*g_ex_sigmayy0_in[n3d_index_ex(iz-1,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmayy0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+1,ix, iy)]*c4*g_ex_sigmayy0_in[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+2,ix, iy)]*c3*g_ex_sigmayy0_in[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+3,ix, iy)]*c2*g_ex_sigmayy0_in[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m2[n3d_index_ex(iz+4,ix, iy)]*c1*g_ex_sigmayy0_in[n3d_index_ex(iz+4,ix,iy)]
	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-4)]*c1*g_ex_sigmayz0_in[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-3)]*c2*g_ex_sigmayz0_in[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-2)]*c3*g_ex_sigmayz0_in[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy-1)]*c4*g_ex_sigmayz0_in[n3d_index_ex(iz,ix,iy-1)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmayz0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+1)]*c5*g_ex_sigmayz0_in[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+2)]*c4*g_ex_sigmayz0_in[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+3)]*c3*g_ex_sigmayz0_in[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+4)]*c2*g_ex_sigmayz0_in[n3d_index_ex(iz,ix,iy+4)]	
									- g_ex_m3[n3d_index_ex(iz,ix, iy+5)]*c1*g_ex_sigmayz0_in[n3d_index_ex(iz,ix,iy+5)]							
	

									+ g_ex_m3[n3d_index_ex(iz,ix-4, iy)]*c1*g_ex_sigmaxz0_in[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m3[n3d_index_ex(iz,ix-3, iy)]*c2*g_ex_sigmaxz0_in[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix-2, iy)]*c3*g_ex_sigmaxz0_in[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix-1, iy)]*c4*g_ex_sigmaxz0_in[n3d_index_ex(iz,ix-1,iy)]	
									+ g_ex_m3[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_sigmaxz0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+1, iy)]*c5*g_ex_sigmaxz0_in[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+2, iy)]*c4*g_ex_sigmaxz0_in[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+3, iy)]*c3*g_ex_sigmaxz0_in[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+4, iy)]*c2*g_ex_sigmaxz0_in[n3d_index_ex(iz,ix+4,iy)]	
									- g_ex_m3[n3d_index_ex(iz,ix+5, iy)]*c1*g_ex_sigmaxz0_in[n3d_index_ex(iz,ix+5,iy)]	;						


		

              g_ex_sigmaxx0_out[n3d_index_ex(iz,ix  ,iy)] = g_ex_sigmaxx0_out[n3d_index_ex(iz,ix  , iy)]	+ g_ex_sigmaxx0_in1[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_x[n3d_index_ex(iz,ix-4, iy)]*c1*g_ex_Vx0_in[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m1_x[n3d_index_ex(iz,ix-3, iy)]*c2*g_ex_Vx0_in[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix-2, iy)]*c3*g_ex_Vx0_in[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix-1, iy)]*c4*g_ex_Vx0_in[n3d_index_ex(iz,ix-1,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vx0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix+1, iy)]*c5*g_ex_Vx0_in[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix+2, iy)]*c4*g_ex_Vx0_in[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix+3, iy)]*c3*g_ex_Vx0_in[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix+4, iy)]*c2*g_ex_Vx0_in[n3d_index_ex(iz,ix+4,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix+5, iy)]*c1*g_ex_Vx0_in[n3d_index_ex(iz,ix+5,iy)]	;						

	    
              g_ex_sigmayy0_out[n3d_index_ex(iz,ix  ,iy)] = g_ex_sigmayy0_out[n3d_index_ex(iz,ix  , iy)]	+ g_ex_sigmayy0_in1[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy-4)]*c1*g_ex_Vy0_in[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy-3)]*c2*g_ex_Vy0_in[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy-2)]*c3*g_ex_Vy0_in[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy-1)]*c4*g_ex_Vy0_in[n3d_index_ex(iz,ix,iy-1)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vy0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix, iy+1)]*c5*g_ex_Vy0_in[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix, iy+2)]*c4*g_ex_Vy0_in[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix, iy+3)]*c3*g_ex_Vy0_in[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix, iy+4)]*c2*g_ex_Vy0_in[n3d_index_ex(iz,ix,iy+4)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix, iy+5)]*c1*g_ex_Vy0_in[n3d_index_ex(iz,ix,iy+5)]	;		


              g_ex_sigmazz0_out[n3d_index_ex(iz,ix  ,iy)] = g_ex_sigmazz0_out[n3d_index_ex(iz,ix  , iy)]	+ g_ex_sigmazz0_in1[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_z[n3d_index_ex(iz-4,ix, iy)]*c1*g_ex_Vz0_in[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m1_z[n3d_index_ex(iz-3,ix, iy)]*c2*g_ex_Vz0_in[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz-2,ix, iy)]*c3*g_ex_Vz0_in[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz-1,ix, iy)]*c4*g_ex_Vz0_in[n3d_index_ex(iz-1,ix,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vz0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz+1,ix, iy)]*c5*g_ex_Vz0_in[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz+2,ix, iy)]*c4*g_ex_Vz0_in[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz+3,ix, iy)]*c3*g_ex_Vz0_in[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz+4,ix, iy)]*c2*g_ex_Vz0_in[n3d_index_ex(iz+4,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz+5,ix, iy)]*c1*g_ex_Vz0_in[n3d_index_ex(iz+5,ix,iy)]	;		 
	
	


              g_ex_sigmaxy0_out[n3d_index_ex(iz,ix  ,iy)] = g_ex_sigmaxy0_out[n3d_index_ex(iz,ix  , iy)]	+ g_ex_sigmaxy0_in1[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_y[n3d_index_ex(iz,ix-4, iy)]*c1*g_ex_Vy0_in[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m1_y[n3d_index_ex(iz,ix-3, iy)]*c2*g_ex_Vy0_in[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix-2, iy)]*c3*g_ex_Vy0_in[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix-1, iy)]*c4*g_ex_Vy0_in[n3d_index_ex(iz,ix-1,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vy0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix+1, iy)]*c5*g_ex_Vy0_in[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix+2, iy)]*c4*g_ex_Vy0_in[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix+3, iy)]*c3*g_ex_Vy0_in[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix+4, iy)]*c2*g_ex_Vy0_in[n3d_index_ex(iz,ix+4,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz,ix+5, iy)]*c1*g_ex_Vy0_in[n3d_index_ex(iz,ix+5,iy)]	

	    
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy-4)]*c1*g_ex_Vx0_in[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy-3)]*c2*g_ex_Vx0_in[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy-2)]*c3*g_ex_Vx0_in[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy-1)]*c4*g_ex_Vx0_in[n3d_index_ex(iz,ix,iy-1)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vx0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix, iy+1)]*c5*g_ex_Vx0_in[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix, iy+2)]*c4*g_ex_Vx0_in[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix, iy+3)]*c3*g_ex_Vx0_in[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix, iy+4)]*c2*g_ex_Vx0_in[n3d_index_ex(iz,ix,iy+4)]	
									- g_ex_m1_x[n3d_index_ex(iz,ix, iy+5)]*c1*g_ex_Vx0_in[n3d_index_ex(iz,ix,iy+5)]	;		


              g_ex_sigmaxz0_out[n3d_index_ex(iz,ix  ,iy)] = g_ex_sigmaxz0_out[n3d_index_ex(iz,ix  , iy)]	+ g_ex_sigmaxz0_in1[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_x[n3d_index_ex(iz-4,ix, iy)]*c1*g_ex_Vx0_in[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m1_x[n3d_index_ex(iz-3,ix, iy)]*c2*g_ex_Vx0_in[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz-2,ix, iy)]*c3*g_ex_Vx0_in[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz-1,ix, iy)]*c4*g_ex_Vx0_in[n3d_index_ex(iz-1,ix,iy)]	
									+ g_ex_m1_x[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vx0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz+1,ix, iy)]*c5*g_ex_Vx0_in[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz+2,ix, iy)]*c4*g_ex_Vx0_in[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz+3,ix, iy)]*c3*g_ex_Vx0_in[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz+4,ix, iy)]*c2*g_ex_Vx0_in[n3d_index_ex(iz+4,ix,iy)]	
									- g_ex_m1_x[n3d_index_ex(iz+5,ix, iy)]*c1*g_ex_Vx0_in[n3d_index_ex(iz+5,ix,iy)]	
							
									+ g_ex_m1_z[n3d_index_ex(iz,ix-4, iy)]*c1*g_ex_Vz0_in[n3d_index_ex(iz,ix-4,iy)]		
									+ g_ex_m1_z[n3d_index_ex(iz,ix-3, iy)]*c2*g_ex_Vz0_in[n3d_index_ex(iz,ix-3,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix-2, iy)]*c3*g_ex_Vz0_in[n3d_index_ex(iz,ix-2,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix-1, iy)]*c4*g_ex_Vz0_in[n3d_index_ex(iz,ix-1,iy)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vz0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix+1, iy)]*c5*g_ex_Vz0_in[n3d_index_ex(iz,ix+1,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix+2, iy)]*c4*g_ex_Vz0_in[n3d_index_ex(iz,ix+2,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix+3, iy)]*c3*g_ex_Vz0_in[n3d_index_ex(iz,ix+3,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix+4, iy)]*c2*g_ex_Vz0_in[n3d_index_ex(iz,ix+4,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix+5, iy)]*c1*g_ex_Vz0_in[n3d_index_ex(iz,ix+5,iy)]	;						


              g_ex_sigmayz0_out[n3d_index_ex(iz,ix  ,iy)] = g_ex_sigmayz0_out[n3d_index_ex(iz,ix  , iy)]	+ g_ex_sigmayz0_in1[n3d_index_ex(iz,ix  , iy)] 
									+ g_ex_m1_y[n3d_index_ex(iz-4,ix, iy)]*c1*g_ex_Vy0_in[n3d_index_ex(iz-4,ix,iy)]		
									+ g_ex_m1_y[n3d_index_ex(iz-3,ix, iy)]*c2*g_ex_Vy0_in[n3d_index_ex(iz-3,ix,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz-2,ix, iy)]*c3*g_ex_Vy0_in[n3d_index_ex(iz-2,ix,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz-1,ix, iy)]*c4*g_ex_Vy0_in[n3d_index_ex(iz-1,ix,iy)]	
									+ g_ex_m1_y[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vy0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz+1,ix, iy)]*c5*g_ex_Vy0_in[n3d_index_ex(iz+1,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz+2,ix, iy)]*c4*g_ex_Vy0_in[n3d_index_ex(iz+2,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz+3,ix, iy)]*c3*g_ex_Vy0_in[n3d_index_ex(iz+3,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz+4,ix, iy)]*c2*g_ex_Vy0_in[n3d_index_ex(iz+4,ix,iy)]	
									- g_ex_m1_y[n3d_index_ex(iz+5,ix, iy)]*c1*g_ex_Vy0_in[n3d_index_ex(iz+5,ix,iy)]	
	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy-4)]*c1*g_ex_Vz0_in[n3d_index_ex(iz,ix,iy-4)]		
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy-3)]*c2*g_ex_Vz0_in[n3d_index_ex(iz,ix,iy-3)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy-2)]*c3*g_ex_Vz0_in[n3d_index_ex(iz,ix,iy-2)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy-1)]*c4*g_ex_Vz0_in[n3d_index_ex(iz,ix,iy-1)]	
									+ g_ex_m1_z[n3d_index_ex(iz,ix, iy)]  *c5*g_ex_Vz0_in[n3d_index_ex(iz,ix,iy)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix, iy+1)]*c5*g_ex_Vz0_in[n3d_index_ex(iz,ix,iy+1)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix, iy+2)]*c4*g_ex_Vz0_in[n3d_index_ex(iz,ix,iy+2)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix, iy+3)]*c3*g_ex_Vz0_in[n3d_index_ex(iz,ix,iy+3)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix, iy+4)]*c2*g_ex_Vz0_in[n3d_index_ex(iz,ix,iy+4)]	
									- g_ex_m1_z[n3d_index_ex(iz,ix, iy+5)]*c1*g_ex_Vz0_in[n3d_index_ex(iz,ix,iy+5)]	;		


}



extern "C" void rtm_gpu_func(int ny, int nz, int nx, 
        float *ex_Vy0_in,  float * ex_Vx0_in, float * ex_Vz0_in, float * ex_sigmayy0_in, float *ex_sigmaxx0_in, float * ex_sigmazz0_in, float * ex_sigmaxy0_in, float * ex_sigmaxz0_in, float * ex_sigmayz0_in,//(nz, nx, nt)
        float *ex_Vy0_in1,  float * ex_Vx0_in1, float * ex_Vz0_in1, float * ex_sigmayy0_in1, float *ex_sigmaxx0_in1, float * ex_sigmazz0_in1, float * ex_sigmaxy0_in1, float * ex_sigmaxz0_in1, float * ex_sigmayz0_in1,//(nz, nx, nt)
        float *ex_Vy0_out,  float * ex_Vx0_out, float * ex_Vz0_out, float * ex_sigmayy0_out, float *ex_sigmaxx0_out, float * ex_sigmazz0_out, float * ex_sigmaxy0_out, float * ex_sigmaxz0_out, float * ex_sigmayz0_out,//(nz, nx, nt)
        float * ex_m1_y, float * ex_m1_x,float * ex_m1_z,float * ex_m2, float * ex_m3, float * ex_m2m3,//)//(nz+10,nx+10)
	float * debug, float * gpu_kernel_time)
{	
     	hipError_t err;
	hipEvent_t start1, start2, start3, stop1, stop2, stop3;
	float elapsedTime1 = 0.0f;
	float elapsedTime2 = 0.0f;
	float elapsedTime3 = 0.0f;
	int g_it;


	hipEventCreate(&start1);
	hipEventCreate(&start2);
	hipEventCreate(&start3);
	hipEventCreate(&stop1);
	hipEventCreate(&stop2);
	hipEventCreate(&stop3);
	//time record


	//data copy in 
	hipEventRecord(start1, 0);
     	rtm_gpu_copy_in(ny, nz, nx, 
			ex_Vy0_in, ex_Vx0_in, ex_Vz0_in, ex_sigmayy0_in, ex_sigmaxx0_in, ex_sigmazz0_in, ex_sigmaxy0_in, ex_sigmaxz0_in, ex_sigmayz0_in,
			ex_Vy0_in1, ex_Vx0_in1, ex_Vz0_in1, ex_sigmayy0_in1, ex_sigmaxx0_in1, ex_sigmazz0_in1, ex_sigmaxy0_in1, ex_sigmaxz0_in1, ex_sigmayz0_in1,
			ex_Vy0_out, ex_Vx0_out, ex_Vz0_out, ex_sigmayy0_out, ex_sigmaxx0_out, ex_sigmazz0_out, ex_sigmaxy0_out, ex_sigmaxz0_out, ex_sigmayz0_out,
			ex_m1_y, ex_m1_x, ex_m1_z, ex_m2, ex_m3, ex_m2m3);
	hipEventRecord(stop1, 0);
	
	
	err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error1: %s.\n", hipGetErrorString(err));
	}	
	
	//RTM computing


	dim3 dimGrid(nz/TZ, nx/TX, ny/TY);
	dim3 dimBlock(TZ, TX, TY);


	hipEventRecord(start2, 0);
	
	fprintf(stderr,"GPU Computing ... ...(NZ=%d, NX=%d, NY=%d, TZ=%d, TX=%d, TY=%d)\n", nz, nx, ny, TZ, TX, TY);
	
	for(g_it = 0; g_it < Steps_write_back; g_it++){
		
		fprintf(stderr, "Step %d\n", g_it);
		rtm_gpu_kernel<<<dimGrid, dimBlock>>>(ny, nz, nx,
			g_ex_Vy0_in, g_ex_Vx0_in, g_ex_Vz0_in, g_ex_sigmayy0_in, g_ex_sigmaxx0_in, g_ex_sigmazz0_in, g_ex_sigmaxy0_in, g_ex_sigmaxz0_in, g_ex_sigmayz0_in,
			g_ex_Vy0_in1, g_ex_Vx0_in1, g_ex_Vz0_in1, g_ex_sigmayy0_in1, g_ex_sigmaxx0_in1, g_ex_sigmazz0_in1, g_ex_sigmaxy0_in1, g_ex_sigmaxz0_in1, g_ex_sigmayz0_in1,
			g_ex_Vy0_out, g_ex_Vx0_out, g_ex_Vz0_out, g_ex_sigmayy0_out, g_ex_sigmaxx0_out, g_ex_sigmazz0_out, g_ex_sigmaxy0_out, g_ex_sigmaxz0_out, g_ex_sigmayz0_out,
			g_ex_m1_y, g_ex_m1_x, g_ex_m1_z, g_ex_m2, g_ex_m3, g_ex_m2m3);
			//hipDeviceSynchronize();

		err = hipGetLastError();
		if(hipSuccess != err){
			fprintf(stderr, "Cuda error2: %s.\n", hipGetErrorString(err));
			}
	
		if(g_it<Steps_write_back-1)	rtm_gpu_change_pointer();	
	}
	hipEventRecord(stop2, 0);
	

	//data copy out
	hipEventRecord(start3, 0);
	
	rtm_gpu_copy_out(ny, nz, nx,	
			ex_Vy0_out, ex_Vx0_out, ex_Vz0_out, ex_sigmayy0_out, ex_sigmaxx0_out, ex_sigmazz0_out, ex_sigmaxy0_out, ex_sigmaxz0_out, ex_sigmayz0_out);
	hipEventRecord(stop3, 0);

	err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error3: %s.\n", hipGetErrorString(err));
	}	


	//hipEventRecord(stop, 0);

	hipEventSynchronize(stop1);
	hipEventSynchronize(stop2);
	hipEventSynchronize(stop3);
	hipEventElapsedTime(&elapsedTime1, start1, stop1);
	hipEventElapsedTime(&elapsedTime2, start2, stop2);
	hipEventElapsedTime(&elapsedTime3, start3, stop3);

	gpu_kernel_time[0] = (float)(elapsedTime1/1000.);
	gpu_kernel_time[1] = (float)(elapsedTime2/1000.);
	gpu_kernel_time[2] = (float)(elapsedTime3/1000.);

	
	fprintf(stderr, "GPU copy in Time: %.4f\n", (float)elapsedTime1/1000.);
	fprintf(stderr, "GPU Comput. Time: %.4f\n", (float)elapsedTime2/1000.);
	fprintf(stderr, "GPU copy ot Time: %.4f\n", (float)elapsedTime3/1000.);

}


__global__ void rtm_gpu_kernel_all_shared(int it,int nt, int nz, int nx,
        float * g_ex_Vx0, float * g_ex_Vz0, float * g_ex_sigmaxx0, float * g_ex_sigmazz0, float * g_ex_sigmaxz0, //(nz, nx, nt)
        float * g_ex_m1_x,float * g_ex_m1_z,float * g_ex_aux_m2_c, float * g_ex_aux_m3_c, float * g_ex_aux_m2m3_c)//(nz+10,	nx+10)
{

	float c1=35.0/294912.0,c2=-405.0/229376.0,c3=567.0/40960.0,c4=-735.0/8192.0,c5=19845.0/16384.0;

	//GPU thread index
	int iz, ix;
	iz = blockIdx.x*blockDim.x + threadIdx.x;
	ix = blockIdx.y*blockDim.y + threadIdx.y;
	//gt = it;
 	
	__shared__ float sh_ex_aux_m2m3_c[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_aux_m2_c[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_aux_m3_c[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_m1_x[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_m1_z[(TZ+10)*(TX+10)];


	__shared__ float sh_ex_Vx0[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_Vz0[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_sigmaxx0[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_sigmazz0[(TZ+10)*(TX+10)];
	__shared__ float sh_ex_sigmaxz0[(TZ+10)*(TX+10)];

	//sh_ex_aux_m2m3_c[threadIdx][];

	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_aux_m2m3_c[index_ex(iz,ix)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_aux_m2_c[index_ex(iz,ix)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_aux_m3_c[index_ex(iz,ix)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_m1_x[index_ex(iz,ix)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_m1_z[index_ex(iz,ix)];

	sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_Vx0[index3d_ex(iz,ix,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_Vz0[index3d_ex(iz,ix,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_sigmaxx0[index3d_ex(iz,ix,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_sigmazz0[index3d_ex(iz,ix,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y)] = g_ex_sigmaxz0[index3d_ex(iz,ix,it+1)];


	if(threadIdx.x<5){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_aux_m2m3_c[index_ex(iz-5,ix)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_aux_m2_c[index_ex(iz-5,ix)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_aux_m3_c[index_ex(iz-5,ix)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_m1_x[index_ex(iz-5,ix)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_m1_z[index_ex(iz-5,ix)];

	sh_ex_Vx0[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_Vx0[index3d_ex(iz-5,ix,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_Vz0[index3d_ex(iz-5,ix,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_sigmaxx0[index3d_ex(iz-5,ix,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_sigmazz0[index3d_ex(iz-5,ix,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x-5,threadIdx.y)] = g_ex_sigmaxz0[index3d_ex(iz-5,ix,it+1)];
	}

	if(threadIdx.x>=TZ-5){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_aux_m2m3_c[index_ex(iz+5,ix)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_aux_m2_c[index_ex(iz+5,ix)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_aux_m3_c[index_ex(iz+5,ix)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_m1_x[index_ex(iz+5,ix)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_m1_z[index_ex(iz+5,ix)];
	
	sh_ex_Vx0[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_Vx0[index3d_ex(iz+5,ix,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_Vz0[index3d_ex(iz+5,ix,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_sigmaxx0[index3d_ex(iz+5,ix,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_sigmazz0[index3d_ex(iz+5,ix,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+5,threadIdx.y)] = g_ex_sigmaxz0[index3d_ex(iz+5,ix,it+1)];
	}
	

	if(threadIdx.y<5){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_aux_m2m3_c[index_ex(iz,ix-5)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_aux_m2_c[index_ex(iz,ix-5)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_aux_m3_c[index_ex(iz,ix-5)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_m1_x[index_ex(iz,ix-5)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_m1_z[index_ex(iz,ix-5)];

	sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_Vx0[index3d_ex(iz,ix-5,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_Vz0[index3d_ex(iz,ix-5,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_sigmaxx0[index3d_ex(iz,ix-5,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_sigmazz0[index3d_ex(iz,ix-5,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y-5)] = g_ex_sigmaxz0[index3d_ex(iz,ix-5,it+1)];
	}


	if(threadIdx.y>=TX-5){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_aux_m2m3_c[index_ex(iz,ix+5)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_aux_m2_c[index_ex(iz,ix+5)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_aux_m3_c[index_ex(iz,ix+5)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_m1_x[index_ex(iz,ix+5)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_m1_z[index_ex(iz,ix+5)];

	sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_Vx0[index3d_ex(iz,ix+5,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_Vz0[index3d_ex(iz,ix+5,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_sigmaxx0[index3d_ex(iz,ix+5,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_sigmazz0[index3d_ex(iz,ix+5,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y+5)] = g_ex_sigmaxz0[index3d_ex(iz,ix+5,it+1)];
	}



	if(threadIdx.x <5 && threadIdx.y <5){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_aux_m2m3_c[index_ex(iz-5,ix-5)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_aux_m2_c[index_ex(iz-5,ix-5)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_aux_m3_c[index_ex(iz-5,ix-5)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_m1_x[index_ex(iz-5,ix-5)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_m1_z[index_ex(iz-5,ix-5)];

	sh_ex_Vx0[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_Vx0[index3d_ex(iz-5,ix-5,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_Vz0[index3d_ex(iz-5,ix-5,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_sigmaxx0[index3d_ex(iz-5,ix-5,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_sigmazz0[index3d_ex(iz-5,ix-5,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x-5,threadIdx.y-5)] = g_ex_sigmaxz0[index3d_ex(iz-5,ix-5,it+1)];
	}

	if(threadIdx.x >= 5+TZ && threadIdx.y >= 5+TX){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_aux_m2m3_c[index_ex(iz+5,ix+5)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_aux_m2_c[index_ex(iz+5,ix+5)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_aux_m3_c[index_ex(iz+5,ix+5)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_m1_x[index_ex(iz+5,ix+5)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_m1_z[index_ex(iz+5,ix+5)];

	sh_ex_Vx0[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_Vx0[index3d_ex(iz+5,ix+5,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_Vz0[index3d_ex(iz+5,ix+5,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_sigmaxx0[index3d_ex(iz+5,ix+5,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_sigmazz0[index3d_ex(iz+5,ix+5,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+5,threadIdx.y+5)] = g_ex_sigmaxz0[index3d_ex(iz+5,ix+5,it+1)];
	}


	if(threadIdx.x >= TZ+5 && threadIdx.y <5){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_aux_m2m3_c[index_ex(iz+5,ix-5)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_aux_m2_c[index_ex(iz+5,ix-5)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_aux_m3_c[index_ex(iz+5,ix-5)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_m1_x[index_ex(iz+5,ix-5)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_m1_z[index_ex(iz+5,ix-5)];
	
	sh_ex_Vx0[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_Vx0[index3d_ex(iz+5,ix-5,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_Vz0[index3d_ex(iz+5,ix-5,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_sigmaxx0[index3d_ex(iz+5,ix-5,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_sigmazz0[index3d_ex(iz+5,ix-5,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+5,threadIdx.y-5)] = g_ex_sigmaxz0[index3d_ex(iz+5,ix-5,it+1)];
	}


	if(threadIdx.x <5 && threadIdx.y >= TX-5){
	sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_aux_m2m3_c[index_ex(iz-5,ix+5)];
	sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_aux_m2_c[index_ex(iz-5,ix+5)];
	sh_ex_aux_m3_c[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_aux_m3_c[index_ex(iz-5,ix+5)];
	sh_ex_m1_x[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_m1_x[index_ex(iz-5,ix+5)];
	sh_ex_m1_z[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_m1_z[index_ex(iz-5,ix+5)];

	sh_ex_Vx0[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_Vx0[index3d_ex(iz-5,ix+5,it+1)];
	sh_ex_Vz0[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_Vz0[index3d_ex(iz-5,ix+5,it+1)];
	sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_sigmaxx0[index3d_ex(iz-5,ix+5,it+1)];
	sh_ex_sigmazz0[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_sigmazz0[index3d_ex(iz-5,ix+5,it+1)];
	sh_ex_sigmaxz0[index_blk_ex(threadIdx.x-5,threadIdx.y+5)] = g_ex_sigmaxz0[index3d_ex(iz-5,ix+5,it+1)];
	}



	__syncthreads();

              g_ex_Vx0[index3d_ex(iz,ix  ,it)] = g_ex_Vx0[index3d_ex(iz,ix  ,it)]	+ g_ex_Vx0[index3d_ex(iz, ix, it+2)]
									+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y-5)]*c1*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y-5)]							
							 		+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y-4)]*c2*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y-4)]		
									+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y-3)]*c3*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y-3)]	
									+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y-2)]*c4*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y-2)]	
									+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y-1)]*c5*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y-1)]	
									- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y)]  *c5*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y)]	
									- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y+1)]*c4*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y+1)]	
									- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y+2)]*c3*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y+2)]	
									- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y+3)]*c2*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y+3)]	
									- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,threadIdx.y+4)]*c1*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y+4)]


									+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y-5)]*c1*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y-5)]							
							 		+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y-4)]*c2*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y-4)]		
									+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y-3)]*c3*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y-3)]	
									+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y-2)]*c4*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y-2)]	
									+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y-1)]*c5*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y-1)]	
									- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y)]  *c5*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y)]	
									- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y+1)]*c4*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y+1)]	
									- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y+2)]*c3*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y+2)]	
									- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y+3)]*c2*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y+3)]	
									- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,threadIdx.y+4)]*c1*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y+4)]	
	


									+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x-4,threadIdx.y)]*c1*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x-4,threadIdx.y)]		
									+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x-3,threadIdx.y)]*c2*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x-3,threadIdx.y)]	
									+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x-2,threadIdx.y)]*c3*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x-2,threadIdx.y)]	
									+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x-1,threadIdx.y)]*c4*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x-1,threadIdx.y)]	
									+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,  threadIdx.y)]  *c5*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y)]	
									- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+1,threadIdx.y)]*c5*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+1,threadIdx.y)]	
									- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+2,threadIdx.y)]*c4*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+2,threadIdx.y)]	
									- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+3,threadIdx.y)]*c3*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+3,threadIdx.y)]	
									- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+4,threadIdx.y)]*c2*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+4,threadIdx.y)]	
									- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x+5,threadIdx.y)]*c1*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x+5,threadIdx.y)]	;						

 
     __syncthreads();

            g_ex_Vz0[index3d_ex(iz,ix  ,it)] = g_ex_Vz0[index3d_ex(iz,ix,  it)]  	+ g_ex_Vz0[index3d_ex(iz,ix  ,it+2)] 
	     								+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-5,threadIdx.y)]*c1*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-5,threadIdx.y)]							
	     						 		+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-4,threadIdx.y)]*c2*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-4,threadIdx.y)]		
	     								+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-3,threadIdx.y)]*c3*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-3,threadIdx.y)]	
	     								+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-2,threadIdx.y)]*c4*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-2,threadIdx.y)]	
	     								+ sh_ex_aux_m2_c[index_blk_ex(threadIdx.x-1,threadIdx.y)]*c5*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x-1,threadIdx.y)]	
	     								- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x,  threadIdx.y)]  *c5*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x,threadIdx.y)]	
	     								- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x+1,threadIdx.y)]*c4*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x+1,threadIdx.y)]	
	     								- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x+2,threadIdx.y)]*c3*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x+2,threadIdx.y)]	
	     								- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x+3,threadIdx.y)]*c2*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x+3,threadIdx.y)]	
	     								- sh_ex_aux_m2_c[index_blk_ex(threadIdx.x+4,threadIdx.y)]*c1*sh_ex_sigmaxx0[index_blk_ex(threadIdx.x+4,threadIdx.y)]	
	     
	
	             							+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-5,threadIdx.y)]*c1*sh_ex_sigmazz0[index_blk_ex(threadIdx.x-5,threadIdx.y)]							
	     						 		+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-4,threadIdx.y)]*c2*sh_ex_sigmazz0[index_blk_ex(threadIdx.x-4,threadIdx.y)]		
	     								+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-3,threadIdx.y)]*c3*sh_ex_sigmazz0[index_blk_ex(threadIdx.x-3,threadIdx.y)]	
	     								+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-2,threadIdx.y)]*c4*sh_ex_sigmazz0[index_blk_ex(threadIdx.x-2,threadIdx.y)]	
	     								+ sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x-1,threadIdx.y)]*c5*sh_ex_sigmazz0[index_blk_ex(threadIdx.x-1,threadIdx.y)]	
	     								- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x,  threadIdx.y)]  *c5*sh_ex_sigmazz0[index_blk_ex(threadIdx.x,threadIdx.y)]	
	     								- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x+1,threadIdx.y)]*c4*sh_ex_sigmazz0[index_blk_ex(threadIdx.x+1,threadIdx.y)]	
	     								- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x+2,threadIdx.y)]*c3*sh_ex_sigmazz0[index_blk_ex(threadIdx.x+2,threadIdx.y)]	
	     								- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x+3,threadIdx.y)]*c2*sh_ex_sigmazz0[index_blk_ex(threadIdx.x+3,threadIdx.y)]	
	     								- sh_ex_aux_m2m3_c[index_blk_ex(threadIdx.x+4,threadIdx.y)]*c1*sh_ex_sigmazz0[index_blk_ex(threadIdx.x+4,threadIdx.y)]	
	     
	     								+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y-4)]*c1*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y-4)]		
	     								+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y-3)]*c2*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y-3)]	
	     								+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y-2)]*c3*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y-2)]	
	     								+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y-1)]*c4*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y-1)]	
	     								+ sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y)]  *c5*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y)]	
	     								- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y+1)]*c5*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y+1)]	
	     								- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y+2)]*c4*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y+2)]	
	     								- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y+3)]*c3*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y+3)]	
	     								- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y+4)]*c2*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y+4)]	
	     								- sh_ex_aux_m3_c[index_blk_ex(threadIdx.x,threadIdx.y+5)]*c1*sh_ex_sigmaxz0[index_blk_ex(threadIdx.x,threadIdx.y+5)]	;							
	


              g_ex_sigmaxx0[index3d_ex(iz,ix  ,it)] = g_ex_sigmaxx0[index3d_ex(iz,ix  ,it)]	+ g_ex_sigmaxx0[index3d_ex(iz,ix  ,it+2)] 
        									+ sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y-4)]*c1*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y-4)]		
        									+ sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y-3)]*c2*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y-3)]	
        									+ sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y-2)]*c3*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y-2)]	
        									+ sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y-1)]*c4*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y-1)]	
        									+ sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y)]  *c5*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y)]	
        									- sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y+1)]*c5*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y+1)]	
        									- sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y+2)]*c4*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y+2)]	
        									- sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y+3)]*c3*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y+3)]	
        									- sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y+4)]*c2*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y+4)]	
        									- sh_ex_m1_x[index_blk_ex(threadIdx.x,threadIdx.y+5)]*c1*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y+5)]	;						
 
    __syncthreads();
             g_ex_sigmazz0[index3d_ex(iz,ix  ,it)] = g_ex_sigmazz0[index3d_ex(iz,ix  ,it)]	+ g_ex_sigmazz0[index3d_ex(iz,ix  ,it+2)] 
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x-4,threadIdx.y)]*c1*sh_ex_Vz0[index_blk_ex(threadIdx.x-4,threadIdx.y)]		
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x-3,threadIdx.y)]*c2*sh_ex_Vz0[index_blk_ex(threadIdx.x-3,threadIdx.y)]	
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x-2,threadIdx.y)]*c3*sh_ex_Vz0[index_blk_ex(threadIdx.x-2,threadIdx.y)]	
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x-1,threadIdx.y)]*c4*sh_ex_Vz0[index_blk_ex(threadIdx.x-1,threadIdx.y)]	
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x,  threadIdx.y)]  *c5*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x+1,threadIdx.y)]*c5*sh_ex_Vz0[index_blk_ex(threadIdx.x+1,threadIdx.y)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x+2,threadIdx.y)]*c4*sh_ex_Vz0[index_blk_ex(threadIdx.x+2,threadIdx.y)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x+3,threadIdx.y)]*c3*sh_ex_Vz0[index_blk_ex(threadIdx.x+3,threadIdx.y)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x+4,threadIdx.y)]*c2*sh_ex_Vz0[index_blk_ex(threadIdx.x+4,threadIdx.y)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x+5,threadIdx.y)]*c1*sh_ex_Vz0[index_blk_ex(threadIdx.x+5,threadIdx.y)]	;						
     __syncthreads();
     g_ex_sigmaxz0[index3d_ex(iz,ix  ,it)] = g_ex_sigmaxz0[index3d_ex(iz,ix  ,it)]	+ g_ex_sigmaxz0[index3d_ex(iz,ix  ,it+2)]	 
										+ sh_ex_m1_x[index_blk_ex(threadIdx.x-5,threadIdx.y)]*c1*sh_ex_Vx0[index_blk_ex(threadIdx.x-5,threadIdx.y)]							
							 			+ sh_ex_m1_x[index_blk_ex(threadIdx.x-4,threadIdx.y)]*c2*sh_ex_Vx0[index_blk_ex(threadIdx.x-4,threadIdx.y)]		
										+ sh_ex_m1_x[index_blk_ex(threadIdx.x-3,threadIdx.y)]*c3*sh_ex_Vx0[index_blk_ex(threadIdx.x-3,threadIdx.y)]	
										+ sh_ex_m1_x[index_blk_ex(threadIdx.x-2,threadIdx.y)]*c4*sh_ex_Vx0[index_blk_ex(threadIdx.x-2,threadIdx.y)]	
										+ sh_ex_m1_x[index_blk_ex(threadIdx.x-1,threadIdx.y)]*c5*sh_ex_Vx0[index_blk_ex(threadIdx.x-1,threadIdx.y)]	
										- sh_ex_m1_x[index_blk_ex(threadIdx.x,  threadIdx.y)]  *c5*sh_ex_Vx0[index_blk_ex(threadIdx.x,threadIdx.y)]	
										- sh_ex_m1_x[index_blk_ex(threadIdx.x+1,threadIdx.y)]*c4*sh_ex_Vx0[index_blk_ex(threadIdx.x+1,threadIdx.y)]	
										- sh_ex_m1_x[index_blk_ex(threadIdx.x+2,threadIdx.y)]*c3*sh_ex_Vx0[index_blk_ex(threadIdx.x+2,threadIdx.y)]	
										- sh_ex_m1_x[index_blk_ex(threadIdx.x+3,threadIdx.y)]*c2*sh_ex_Vx0[index_blk_ex(threadIdx.x+3,threadIdx.y)]	
										- sh_ex_m1_x[index_blk_ex(threadIdx.x+4,threadIdx.y)]*c1*sh_ex_Vx0[index_blk_ex(threadIdx.x+4,threadIdx.y)]	//;
	
        
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y-5)]*c1*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y-5)]							
							 			+ sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y-4)]*c2*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y-4)]		
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y-3)]*c3*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y-3)]	
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y-2)]*c4*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y-2)]	
										+ sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y-1)]*c5*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y-1)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y)]  *c5*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y+1)]*c4*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y+1)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y+2)]*c3*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y+2)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y+3)]*c2*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y+3)]	
										- sh_ex_m1_z[index_blk_ex(threadIdx.x,threadIdx.y+4)]*c1*sh_ex_Vz0[index_blk_ex(threadIdx.x,threadIdx.y+4)]	;
		
	__syncthreads();


	}


__global__ void rtm_gpu_kernel_l1(int it,int nt, int nz, int nx,
        float * g_ex_Vx0, float * g_ex_Vz0, float * g_ex_sigmaxx0, float * g_ex_sigmazz0, float * g_ex_sigmaxz0, //(nz, nx, nt)
        float * g_ex_m1_x,float * g_ex_m1_z,float * g_ex_aux_m2_c, float * g_ex_aux_m3_c, float * g_ex_aux_m2m3_c)//(nz+10,	nx+10)
{

	float c1=35.0/294912.0,c2=-405.0/229376.0,c3=567.0/40960.0,c4=-735.0/8192.0,c5=19845.0/16384.0;

	//GPU thread index
	int iz, ix;
	iz = blockIdx.x*blockDim.x + threadIdx.x;
	ix = blockIdx.y*blockDim.y + threadIdx.y;
	//gt = it;
 	
              g_ex_Vx0[index3d_ex(iz,ix  ,it)] = g_ex_Vx0[index3d_ex(iz,ix  ,it)]	+ g_ex_Vx0[index3d_ex(iz, ix, it+2)]
									+ g_ex_aux_m2m3_c[index_ex(iz,ix-5)]*c1*g_ex_sigmaxx0[index3d_ex(iz,ix-5,it+1)]							
							 		+ g_ex_aux_m2m3_c[index_ex(iz,ix-4)]*c2*g_ex_sigmaxx0[index3d_ex(iz,ix-4,it+1)]		
									+ g_ex_aux_m2m3_c[index_ex(iz,ix-3)]*c3*g_ex_sigmaxx0[index3d_ex(iz,ix-3,it+1)]	
									+ g_ex_aux_m2m3_c[index_ex(iz,ix-2)]*c4*g_ex_sigmaxx0[index3d_ex(iz,ix-2,it+1)]	
									+ g_ex_aux_m2m3_c[index_ex(iz,ix-1)]*c5*g_ex_sigmaxx0[index3d_ex(iz,ix-1,it+1)]	
									- g_ex_aux_m2m3_c[index_ex(iz,ix)]  *c5*g_ex_sigmaxx0[index3d_ex(iz,ix,it+1)]	
									- g_ex_aux_m2m3_c[index_ex(iz,ix+1)]*c4*g_ex_sigmaxx0[index3d_ex(iz,ix+1,it+1)]	
									- g_ex_aux_m2m3_c[index_ex(iz,ix+2)]*c3*g_ex_sigmaxx0[index3d_ex(iz,ix+2,it+1)]	
									- g_ex_aux_m2m3_c[index_ex(iz,ix+3)]*c2*g_ex_sigmaxx0[index3d_ex(iz,ix+3,it+1)]	
									- g_ex_aux_m2m3_c[index_ex(iz,ix+4)]*c1*g_ex_sigmaxx0[index3d_ex(iz,ix+4,it+1)]


									+ g_ex_aux_m2_c[index_ex(iz,ix-5)]*c1*g_ex_sigmazz0[index3d_ex(iz,ix-5,it+1)]							
							 		+ g_ex_aux_m2_c[index_ex(iz,ix-4)]*c2*g_ex_sigmazz0[index3d_ex(iz,ix-4,it+1)]		
									+ g_ex_aux_m2_c[index_ex(iz,ix-3)]*c3*g_ex_sigmazz0[index3d_ex(iz,ix-3,it+1)]	
									+ g_ex_aux_m2_c[index_ex(iz,ix-2)]*c4*g_ex_sigmazz0[index3d_ex(iz,ix-2,it+1)]	
									+ g_ex_aux_m2_c[index_ex(iz,ix-1)]*c5*g_ex_sigmazz0[index3d_ex(iz,ix-1,it+1)]	
									- g_ex_aux_m2_c[index_ex(iz,ix)]  *c5*g_ex_sigmazz0[index3d_ex(iz,ix,it+1)]	
									- g_ex_aux_m2_c[index_ex(iz,ix+1)]*c4*g_ex_sigmazz0[index3d_ex(iz,ix+1,it+1)]	
									- g_ex_aux_m2_c[index_ex(iz,ix+2)]*c3*g_ex_sigmazz0[index3d_ex(iz,ix+2,it+1)]	
									- g_ex_aux_m2_c[index_ex(iz,ix+3)]*c2*g_ex_sigmazz0[index3d_ex(iz,ix+3,it+1)]	
									- g_ex_aux_m2_c[index_ex(iz,ix+4)]*c1*g_ex_sigmazz0[index3d_ex(iz,ix+4,it+1)]	
	


									+ g_ex_aux_m3_c[index_ex(iz-4,ix)]*c1*g_ex_sigmaxz0[index3d_ex(iz-4,ix,it+1)]		
									+ g_ex_aux_m3_c[index_ex(iz-3,ix)]*c2*g_ex_sigmaxz0[index3d_ex(iz-3,ix,it+1)]	
									+ g_ex_aux_m3_c[index_ex(iz-2,ix)]*c3*g_ex_sigmaxz0[index3d_ex(iz-2,ix,it+1)]	
									+ g_ex_aux_m3_c[index_ex(iz-1,ix)]*c4*g_ex_sigmaxz0[index3d_ex(iz-1,ix,it+1)]	
									+ g_ex_aux_m3_c[index_ex(iz,ix)]  *c5*g_ex_sigmaxz0[index3d_ex(iz,ix,it+1)]	
									- g_ex_aux_m3_c[index_ex(iz+1,ix)]*c5*g_ex_sigmaxz0[index3d_ex(iz+1,ix,it+1)]	
									- g_ex_aux_m3_c[index_ex(iz+2,ix)]*c4*g_ex_sigmaxz0[index3d_ex(iz+2,ix,it+1)]	
									- g_ex_aux_m3_c[index_ex(iz+3,ix)]*c3*g_ex_sigmaxz0[index3d_ex(iz+3,ix,it+1)]	
									- g_ex_aux_m3_c[index_ex(iz+4,ix)]*c2*g_ex_sigmaxz0[index3d_ex(iz+4,ix,it+1)]	
									- g_ex_aux_m3_c[index_ex(iz+5,ix)]*c1*g_ex_sigmaxz0[index3d_ex(iz+5,ix,it+1)]	;						

 

            g_ex_Vz0[index3d_ex(iz,ix  ,it)] = g_ex_Vz0[index3d_ex(iz,ix  ,it)]  	+ g_ex_Vz0[index3d_ex(iz,ix  ,it+2)] 
	     								+ g_ex_aux_m2_c[index_ex(iz-5,ix)]*c1*g_ex_sigmaxx0[index3d_ex(iz-5,ix,it+1)]							
	     						 		+ g_ex_aux_m2_c[index_ex(iz-4,ix)]*c2*g_ex_sigmaxx0[index3d_ex(iz-4,ix,it+1)]		
	     								+ g_ex_aux_m2_c[index_ex(iz-3,ix)]*c3*g_ex_sigmaxx0[index3d_ex(iz-3,ix,it+1)]	
	     								+ g_ex_aux_m2_c[index_ex(iz-2,ix)]*c4*g_ex_sigmaxx0[index3d_ex(iz-2,ix,it+1)]	
	     								+ g_ex_aux_m2_c[index_ex(iz-1,ix)]*c5*g_ex_sigmaxx0[index3d_ex(iz-1,ix,it+1)]	
	     								- g_ex_aux_m2_c[index_ex(iz,ix)]  *c5*g_ex_sigmaxx0[index3d_ex(iz,ix,it+1)]	
	     								- g_ex_aux_m2_c[index_ex(iz+1,ix)]*c4*g_ex_sigmaxx0[index3d_ex(iz+1,ix,it+1)]	
	     								- g_ex_aux_m2_c[index_ex(iz+2,ix)]*c3*g_ex_sigmaxx0[index3d_ex(iz+2,ix,it+1)]	
	     								- g_ex_aux_m2_c[index_ex(iz+3,ix)]*c2*g_ex_sigmaxx0[index3d_ex(iz+3,ix,it+1)]	
	     								- g_ex_aux_m2_c[index_ex(iz+4,ix)]*c1*g_ex_sigmaxx0[index3d_ex(iz+4,ix,it+1)]	
	     
	
	             							+ g_ex_aux_m2m3_c[index_ex(iz-5,ix)]*c1*g_ex_sigmazz0[index3d_ex(iz-5,ix,it+1)]							
	     						 		+ g_ex_aux_m2m3_c[index_ex(iz-4,ix)]*c2*g_ex_sigmazz0[index3d_ex(iz-4,ix,it+1)]		
	     								+ g_ex_aux_m2m3_c[index_ex(iz-3,ix)]*c3*g_ex_sigmazz0[index3d_ex(iz-3,ix,it+1)]	
	     								+ g_ex_aux_m2m3_c[index_ex(iz-2,ix)]*c4*g_ex_sigmazz0[index3d_ex(iz-2,ix,it+1)]	
	     								+ g_ex_aux_m2m3_c[index_ex(iz-1,ix)]*c5*g_ex_sigmazz0[index3d_ex(iz-1,ix,it+1)]	
	     								- g_ex_aux_m2m3_c[index_ex(iz,ix)]  *c5*g_ex_sigmazz0[index3d_ex(iz,ix,it+1)]	
	     								- g_ex_aux_m2m3_c[index_ex(iz+1,ix)]*c4*g_ex_sigmazz0[index3d_ex(iz+1,ix,it+1)]	
	     								- g_ex_aux_m2m3_c[index_ex(iz+2,ix)]*c3*g_ex_sigmazz0[index3d_ex(iz+2,ix,it+1)]	
	     								- g_ex_aux_m2m3_c[index_ex(iz+3,ix)]*c2*g_ex_sigmazz0[index3d_ex(iz+3,ix,it+1)]	
	     								- g_ex_aux_m2m3_c[index_ex(iz+4,ix)]*c1*g_ex_sigmazz0[index3d_ex(iz+4,ix,it+1)]	
	     
	     								+ g_ex_aux_m3_c[index_ex(iz,ix-4)]*c1*g_ex_sigmaxz0[index3d_ex(iz,ix-4,it+1)]		
	     								+ g_ex_aux_m3_c[index_ex(iz,ix-3)]*c2*g_ex_sigmaxz0[index3d_ex(iz,ix-3,it+1)]	
	     								+ g_ex_aux_m3_c[index_ex(iz,ix-2)]*c3*g_ex_sigmaxz0[index3d_ex(iz,ix-2,it+1)]	
	     								+ g_ex_aux_m3_c[index_ex(iz,ix-1)]*c4*g_ex_sigmaxz0[index3d_ex(iz,ix-1,it+1)]	
	     								+ g_ex_aux_m3_c[index_ex(iz,ix)]  *c5*g_ex_sigmaxz0[index3d_ex(iz,ix,it+1)]	
	     								- g_ex_aux_m3_c[index_ex(iz,ix+1)]*c5*g_ex_sigmaxz0[index3d_ex(iz,ix+1,it+1)]	
	     								- g_ex_aux_m3_c[index_ex(iz,ix+2)]*c4*g_ex_sigmaxz0[index3d_ex(iz,ix+2,it+1)]	
	     								- g_ex_aux_m3_c[index_ex(iz,ix+3)]*c3*g_ex_sigmaxz0[index3d_ex(iz,ix+3,it+1)]	
	     								- g_ex_aux_m3_c[index_ex(iz,ix+4)]*c2*g_ex_sigmaxz0[index3d_ex(iz,ix+4,it+1)]	
	     								- g_ex_aux_m3_c[index_ex(iz,ix+5)]*c1*g_ex_sigmaxz0[index3d_ex(iz,ix+5,it+1)]	;							
	


              g_ex_sigmaxx0[index3d_ex(iz,ix  ,it)] = g_ex_sigmaxx0[index3d_ex(iz,ix  ,it)]	+ g_ex_sigmaxx0[index3d_ex(iz,ix  ,it+2)] 
        									+ g_ex_m1_x[index_ex(iz,ix-4)]*c1*g_ex_Vx0[index3d_ex(iz,ix-4,it+1)]		
        									+ g_ex_m1_x[index_ex(iz,ix-3)]*c2*g_ex_Vx0[index3d_ex(iz,ix-3,it+1)]	
        									+ g_ex_m1_x[index_ex(iz,ix-2)]*c3*g_ex_Vx0[index3d_ex(iz,ix-2,it+1)]	
        									+ g_ex_m1_x[index_ex(iz,ix-1)]*c4*g_ex_Vx0[index3d_ex(iz,ix-1,it+1)]	
        									+ g_ex_m1_x[index_ex(iz,ix)]  *c5*g_ex_Vx0[index3d_ex(iz,ix,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+1)]*c5*g_ex_Vx0[index3d_ex(iz,ix+1,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+2)]*c4*g_ex_Vx0[index3d_ex(iz,ix+2,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+3)]*c3*g_ex_Vx0[index3d_ex(iz,ix+3,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+4)]*c2*g_ex_Vx0[index3d_ex(iz,ix+4,it+1)]	
        									- g_ex_m1_x[index_ex(iz,ix+5)]*c1*g_ex_Vx0[index3d_ex(iz,ix+5,it+1)]	;						
 
             g_ex_sigmazz0[index3d_ex(iz,ix  ,it)] = g_ex_sigmazz0[index3d_ex(iz,ix  ,it)]	+ g_ex_sigmazz0[index3d_ex(iz,ix  ,it+2)] 
										+ g_ex_m1_z[index_ex(iz-4,ix)]*c1*g_ex_Vz0[index3d_ex(iz-4,ix,it+1)]		
										+ g_ex_m1_z[index_ex(iz-3,ix)]*c2*g_ex_Vz0[index3d_ex(iz-3,ix,it+1)]	
										+ g_ex_m1_z[index_ex(iz-2,ix)]*c3*g_ex_Vz0[index3d_ex(iz-2,ix,it+1)]	
										+ g_ex_m1_z[index_ex(iz-1,ix)]*c4*g_ex_Vz0[index3d_ex(iz-1,ix,it+1)]	
										+ g_ex_m1_z[index_ex(iz,ix)]  *c5*g_ex_Vz0[index3d_ex(iz,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+1,ix)]*c5*g_ex_Vz0[index3d_ex(iz+1,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+2,ix)]*c4*g_ex_Vz0[index3d_ex(iz+2,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+3,ix)]*c3*g_ex_Vz0[index3d_ex(iz+3,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+4,ix)]*c2*g_ex_Vz0[index3d_ex(iz+4,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz+5,ix)]*c1*g_ex_Vz0[index3d_ex(iz+5,ix,it+1)]	;						
     
	g_ex_sigmaxz0[index3d_ex(iz,ix  ,it)] = g_ex_sigmaxz0[index3d_ex(iz,ix  ,it)]	+ g_ex_sigmaxz0[index3d_ex(iz,ix  ,it+2)]	 
										+ g_ex_m1_x[index_ex(iz-5,ix)]*c1*g_ex_Vx0[index3d_ex(iz-5,ix,it+1)]							
							 			+ g_ex_m1_x[index_ex(iz-4,ix)]*c2*g_ex_Vx0[index3d_ex(iz-4,ix,it+1)]		
										+ g_ex_m1_x[index_ex(iz-3,ix)]*c3*g_ex_Vx0[index3d_ex(iz-3,ix,it+1)]	
										+ g_ex_m1_x[index_ex(iz-2,ix)]*c4*g_ex_Vx0[index3d_ex(iz-2,ix,it+1)]	
										+ g_ex_m1_x[index_ex(iz-1,ix)]*c5*g_ex_Vx0[index3d_ex(iz-1,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz,ix)]  *c5*g_ex_Vx0[index3d_ex(iz,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+1,ix)]*c4*g_ex_Vx0[index3d_ex(iz+1,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+2,ix)]*c3*g_ex_Vx0[index3d_ex(iz+2,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+3,ix)]*c2*g_ex_Vx0[index3d_ex(iz+3,ix,it+1)]	
										- g_ex_m1_x[index_ex(iz+4,ix)]*c1*g_ex_Vx0[index3d_ex(iz+4,ix,it+1)]	//;
	
        
										+ g_ex_m1_z[index_ex(iz,ix-5)]*c1*g_ex_Vz0[index3d_ex(iz,ix-5,it+1)]							
							 			+ g_ex_m1_z[index_ex(iz,ix-4)]*c2*g_ex_Vz0[index3d_ex(iz,ix-4,it+1)]		
										+ g_ex_m1_z[index_ex(iz,ix-3)]*c3*g_ex_Vz0[index3d_ex(iz,ix-3,it+1)]	
										+ g_ex_m1_z[index_ex(iz,ix-2)]*c4*g_ex_Vz0[index3d_ex(iz,ix-2,it+1)]	
										+ g_ex_m1_z[index_ex(iz,ix-1)]*c5*g_ex_Vz0[index3d_ex(iz,ix-1,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix)]  *c5*g_ex_Vz0[index3d_ex(iz,ix,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+1)]*c4*g_ex_Vz0[index3d_ex(iz,ix+1,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+2)]*c3*g_ex_Vz0[index3d_ex(iz,ix+2,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+3)]*c2*g_ex_Vz0[index3d_ex(iz,ix+3,it+1)]	
										- g_ex_m1_z[index_ex(iz,ix+4)]*c1*g_ex_Vz0[index3d_ex(iz,ix+4,it+1)]	;
		

	}



